#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>

#include "merlin/logger.hpp"  // MESSAGE
#include "merlin/array/array.hpp"  // merlin::Array
#include "merlin/array/parcel.hpp"  // merlin::Parcel

// function printing elements of a CUDA tensor
__global__ void print_gpu_tensor(double * gpu_data) {
    CUDAOUT("GPU element at %d is %.1f.\n", (blockIdx.x*blockDim.x+threadIdx.x),
            gpu_data[blockIdx.x*blockDim.x+threadIdx.x]);
}

// function double elements of a CUDA tensor
__global__ void double_element(double * gpu_data) {
    gpu_data[blockIdx.x*blockDim.x+threadIdx.x] *= 2;
}

int main(void) {
    // initialize an tensor
    double A_data[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    std::uint64_t dims[2] = {2, 3};
    std::uint64_t strides[2] = {5*sizeof(double), 2*sizeof(double)};
    merlin::array::Array A(A_data, 2, dims, strides, false);

    // copy data to GPU and print each element of the tensor
    merlin::array::Parcel B(A.shape());
    B.transfer_data_to_gpu(A);
    print_gpu_tensor<<<1,B.size()>>>(B.data());
    hipDeviceSynchronize();

    // expected result [[1,3,5], [6,8,10]]
    MESSAGE("Expected result: 1.0 3.0 5.0 6.0 8.0 10.0\n");

    // clone result to another Parcel
    merlin::array::Parcel C(B);
    // doubling result
    double_element<<<1,C.size()>>>(C.data());
    A.clone_data_from_gpu(C);
    MESSAGE("After doubling, each element of tensor is: ");
    for (merlin::array::Array::iterator it = A.begin(); it != A.end(); it++) {
        std::printf("%.1f ", A[it.index()]);
    }
    std::printf("\n");
}
