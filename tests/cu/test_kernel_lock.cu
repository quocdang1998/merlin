#include "hip/hip_runtime.h"
#include <cstdio>
#include "merlin/device/lock.hpp"

#define NUMBLOCKS  32
#define NUMTHREADS 8

// each thread 0 of block increases the value of count by 1
// data race will result in undefined behavior
__global__ void increment_without_lock(int * count) {
    if (threadIdx.x == 0) {
        *count += 1;
    }
    __syncthreads();
}

// each thread 0 of block increases sequentially the value of count by 1
// result is the number of block provided to the kernel
__global__ void increment_with_lock(merlin::KernelLock lock, int *numBlocks) {
    lock.lock();
    // only one thread block can enter this region at a time
    if (threadIdx.x == 0) {
        numBlocks[0] = numBlocks[0] + 1;
    }
    lock.unlock();
}

int main(void) {
    int count = 0;
    int * gpu_count;
    hipMalloc(&gpu_count, sizeof(int));
    merlin::KernelLock lock;

    // test case without lock (data race)
    hipMemset(gpu_count, 0, sizeof(int));
    increment_without_lock<<<NUMBLOCKS,NUMTHREADS>>>(gpu_count);
    hipMemcpy(&count, gpu_count, sizeof(int), hipMemcpyDeviceToHost);
    std::printf("Counting in the unlocked case: %i\n", count);

    // test case with lock (no data race)
    count = 0;
    hipMemset(gpu_count, 0, sizeof(int));
    increment_with_lock<<<NUMBLOCKS,NUMTHREADS>>>(lock, gpu_count);
    hipMemcpy(&count, gpu_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Counting in the locked case: %i\n", count);

    hipFree(gpu_count);
}
