#include "hip/hip_runtime.h"
#include <cstdio>

#include "merlin/tensor.hpp"

// function printing elements of a CUDA tensor
__global__ void print_gpu_tensor(float * gpu_data) {
    std::printf("GPU element at %d is %f.\n", (blockIdx.x*blockDim.x+threadIdx.x),
                gpu_data[blockIdx.x*blockDim.x+threadIdx.x]);
}

// function double elements of a CUDA tensor
__global__ void double_element(float * gpu_data) {
    gpu_data[blockIdx.x*blockDim.x+threadIdx.x] *= 2;
}

int main(void) {
    // initialize an tensor
    float A_data[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    unsigned int dims[2] = {2, 3};
    unsigned int strides[2] = {5*sizeof(float), 2*sizeof(float)};
    merlin::Tensor A(A_data, 2, dims, strides, false);

    // copy data to GPU and print each element of the tensor
    A.sync_to_gpu();
    print_gpu_tensor<<<1,A.size()>>>(A.gpu_data().back());
    hipDeviceSynchronize();

    // expected result [[1,3,5], [6,8,10]]
    std::printf("Expected result: 1.0 3.0 5.0 6.0 8.0 10.0\n");

    // doubling result
    double_element<<<1,A.size()>>>(A.gpu_data().back());
    A.sync_from_gpu(A.gpu_data().back());
    std::printf("After doubling, each element of tensor is: ");
    for (merlin::Tensor::iterator it = A.begin(); it != A.end(); it++) {
        std::printf("%.1f ", A[it.index()]);
    }
    std::printf("\n");

    // free the result
    A.free_data_from_gpu();
}
