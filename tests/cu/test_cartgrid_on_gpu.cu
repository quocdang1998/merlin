#include "hip/hip_runtime.h"
#include "merlin/splint/cartesian_grid.hpp"
#include "merlin/logger.hpp"
#include "merlin/utils.hpp"

#include <cstdio>

__global__ void print_grid_from_shared_mem(merlin::splint::CartesianGrid * grid_ptr) {
    std::uint64_t thread_idx = merlin::flatten_thread_index(), block_size = merlin::size_of_block();
    extern __shared__ merlin::splint::CartesianGrid share_ptr[];
    grid_ptr->copy_by_block(share_ptr, share_ptr+1, thread_idx, block_size);
    CUDAOUT("Cartesian Grid on GPU (shared mem):\n");
    for (int i = 0; i < share_ptr->ndim(); i++) {
        std::printf("Vector %d:", i);
        for (int j = 0; j < share_ptr->shape()[i]; j++) {
            std::printf(" %.2f", share_ptr->grid_vectors()[i][j]);
        }
        std::printf("\n");
    }
    CUDAOUT("Cartesian Grid size on GPU (shared mem): %lu\n", share_ptr->size());
}

__global__ void print_grid(merlin::splint::CartesianGrid * grid_ptr) {
    CUDAOUT("Cartesian Grid on GPU:\n");
    for (int i = 0; i < grid_ptr->ndim(); i++) {
        std::printf("Vector %d:", i);
        for (int j = 0; j < grid_ptr->shape()[i]; j++) {
            std::printf(" %.2f", grid_ptr->grid_vectors()[i][j]);
        }
        std::printf("\n");
    }
}

int main(void) {
    merlin::Vector<double> v1 = {0.1, 0.2, 0.3};
    merlin::Vector<double> v2 = {1.0, 2.0, 3.0, 4.0};
    merlin::Vector<double> v3 = {0.0, 0.25};
    merlin::splint::CartesianGrid cart_gr({v1, v2, v3});

    merlin::splint::CartesianGrid * gpu_gr;
    hipMalloc(&gpu_gr, cart_gr.cumalloc_size());
    cart_gr.copy_to_gpu(gpu_gr, gpu_gr+1);
    print_grid<<<1,1>>>(gpu_gr);
    print_grid_from_shared_mem<<<1,1,cart_gr.sharedmem_size()>>>(gpu_gr);
    hipFree(gpu_gr);
}
