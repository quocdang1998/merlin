#include "hip/hip_runtime.h"
#include "merlin/grid.hpp"

#include "cstdio"

__global__ void print_grid_from_shared_mem(merlin::CartesianGrid * grid_ptr) {
    CUDAOUT("Cartesian Grid on GPU (shared mem):\n");
    extern __shared__ merlin::CartesianGrid share_ptr[];
    grid_ptr->copy_to_shared_mem(share_ptr, share_ptr+1);
    for (int i = 0; i < share_ptr->ndim(); i++) {
        std::printf("Vector %d:", i);
        for (int j = 0; j < share_ptr->grid_vectors()[i].size(); j++) {
            std::printf(" %.2f", share_ptr->grid_vectors()[i][j]);
        }
        std::printf("\n");
    }
}

__global__ void print_grid(merlin::CartesianGrid * grid_ptr) {
    CUDAOUT("Cartesian Grid on GPU:\n");
    for (int i = 0; i < grid_ptr->ndim(); i++) {
        std::printf("Vector %d:", i);
        for (int j = 0; j < grid_ptr->grid_vectors()[i].size(); j++) {
            std::printf(" %.2f", grid_ptr->grid_vectors()[i][j]);
        }
        std::printf("\n");
    }
}

int main(void) {
    merlin::floatvec v1 = {0.1, 0.2, 0.3};
    merlin::floatvec v2 = {1.0, 2.0, 3.0, 4.0};
    merlin::floatvec v3 = {0.0, 0.25};
    merlin::CartesianGrid cart_gr = {v1, v2, v3};

    merlin::CartesianGrid * gpu_gr;
    hipMalloc(&gpu_gr, cart_gr.malloc_size());
    cart_gr.copy_to_gpu(gpu_gr, gpu_gr+1);
    print_grid<<<1,1>>>(gpu_gr);
    print_grid_from_shared_mem<<<1,1,cart_gr.malloc_size()>>>(gpu_gr);
    hipFree(gpu_gr);
}
