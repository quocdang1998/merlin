#include "hip/hip_runtime.h"
#include "merlin/grid/cartesian_grid.hpp"
#include "merlin/cuda/memory.hpp"
#include "merlin/logger.hpp"
#include "merlin/utils.hpp"
#include "merlin/settings.hpp"

#include <cstdio>

__global__ void print_trivial_copyable(merlin::Index * p_index, double * p_a) {
    std::uint64_t thread_idx = merlin::flatten_thread_index(), block_size = merlin::size_of_block();
    CUDAOUT("Index element: %u\nFloat element %f\n", unsigned((*p_index)[thread_idx]), *p_a);
}

__global__ void print_grid_from_shared_mem(merlin::grid::CartesianGrid * grid_ptr) {
    std::uint64_t thread_idx = merlin::flatten_thread_index(), block_size = merlin::size_of_block();
    extern __shared__ merlin::grid::CartesianGrid share_ptr[];
    grid_ptr->copy_by_block(share_ptr, share_ptr+1, thread_idx, block_size);
    CUDAOUT("Cartesian Grid on GPU (shared mem):\n");
    for (int i = 0; i < share_ptr->ndim(); i++) {
        std::printf("Vector %d:", i);
        for (int j = 0; j < share_ptr->shape()[i]; j++) {
            std::printf(" %.2f", share_ptr->grid_vectors()[i][j]);
        }
        std::printf("\n");
    }
    CUDAOUT("Cartesian Grid size on GPU (shared mem): %lu\n", share_ptr->size());
}

__global__ void print_grid(merlin::grid::CartesianGrid * grid_ptr) {
    CUDAOUT("Cartesian Grid on GPU (ndim = %u):\n", unsigned(grid_ptr->ndim()));
    for (int i = 0; i < grid_ptr->ndim(); i++) {
        std::printf("Vector %d:", i);
        for (int j = 0; j < grid_ptr->shape()[i]; j++) {
            std::printf(" %.2f", grid_ptr->grid_vectors()[i][j]);
        }
        std::printf("\n");
    }
}

int main(void) {
    merlin::Vector<double> v1 = {0.1, 0.2, 0.3};
    merlin::Vector<double> v2 = {1.0, 2.0, 3.0, 4.0};
    merlin::Vector<double> v3 = {0.0, 0.25};
    merlin::grid::CartesianGrid cart_gr({v1, v2, v3});
    merlin::Index random_array;
    random_array.fill(5);
    double a = 7;

    merlin::cuda::Memory mem(0, cart_gr, random_array, a);

    merlin::grid::CartesianGrid * gpu_gr = mem.get<0>();
    print_grid<<<1,1>>>(gpu_gr);
    print_grid_from_shared_mem<<<1,1,cart_gr.sharedmem_size()>>>(gpu_gr);

    print_trivial_copyable<<<1, 2>>>(mem.get<1>(), mem.get<2>());
}
