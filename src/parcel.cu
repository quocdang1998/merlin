// Copyright 2022 quocdang1998
#include "merlin/parcel.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array.hpp"  // merlin::Array
#include "merlin/logger.hpp"  // FAILURE
#include "merlin/utils.hpp"  // merlin::inner_prod, merlin::contiguous_strides,
                             // merlin::get_current_device, merlin::contiguous_to_ndim_idx
                             // merlin::array_copy
#include "merlin/vector.hpp"  // merlin::intvec

namespace merlin {

// Default constructor
Parcel::Parcel(void) {}

// Constructor from CPU array
Parcel::Parcel(const Array & cpu_array, std::uintptr_t stream) : NdData(cpu_array) {
    // get device id
    hipGetDevice(&(this->device_id_));
    // allocate data
    hipError_t err_ = hipMalloc(&(this->data_), sizeof(float) * this->size());
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // cast stream
    hipStream_t copy_stream = reinterpret_cast<hipStream_t>(stream);
    // reset strides vector
    this->strides_ = contiguous_strides(this->shape_, sizeof(float));
    // create copy function
    auto copy_func = std::bind(hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               hipMemcpyHostToDevice, copy_stream);
    // copy data to GPU
    array_copy(dynamic_cast<NdData *>(this), dynamic_cast<const NdData *>(&cpu_array), copy_func);
}

// Check if current device holds data pointed by object
int Parcel::check_device(void) const {
    return (this->device_id_ - get_current_device());
}

// Copy constructor
Parcel::Parcel(const Parcel & src) : NdData(src) {
    // get device id
    hipGetDevice(&(this->device_id_));
    // reform strides vector
    this->strides_ = contiguous_strides(this->shape_, sizeof(float));
    // allocate data
    hipError_t err_ = hipMalloc(&(this->data_), sizeof(float) * this->size());
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // create copy function
    auto copy_func = std::bind(hipMemcpyPeer, std::placeholders::_1, this->device_id_,
                               std::placeholders::_2, src.device_id_, std::placeholders::_3);
    // copy data to GPU
    array_copy(dynamic_cast<NdData *>(this), dynamic_cast<const NdData *>(&src), copy_func);
}

// Copy assignement
Parcel & Parcel::operator=(const Parcel & src) {
    // free old data
    this->free_current_data();
    // copy metadata and reform strides vector
    this->NdData::operator=(src);
    this->strides_ = contiguous_strides(this->shape_, sizeof(float));
    // allocate data
    hipError_t err_ = hipMalloc(&(this->data_), sizeof(float) * this->size());
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // create copy function
    auto copy_func = std::bind(hipMemcpyPeer, std::placeholders::_1, this->device_id_,
                               std::placeholders::_2, src.device_id_, std::placeholders::_3);
    // copy data to GPU
    array_copy(dynamic_cast<NdData *>(this), dynamic_cast<const NdData *>(&src), copy_func);
    return *this;
}

// Move constructor
Parcel::Parcel(Parcel && src) : NdData(src) {
    // move device id
    this->device_id_ = src.device_id_;
    // take over pointer to source
    src.data_ = NULL;
}

// Move assignment
Parcel & Parcel::operator=(Parcel && src) {
    // free old data
    this->free_current_data();
    // move device id
    this->device_id_ = src.device_id_;
    // copy metadata
    this->NdData::operator=(src);
    // take over pointer to source
    src.data_ = NULL;
    return *this;
}

// Copy data to a pre-allocated memory
void Parcel::copy_to_gpu(Parcel * gpu_ptr, std::uint64_t * shape_strides_ptr) {
    // initialize buffer to store data of the copy before cloning it to GPU
    Parcel copy_on_gpu;
    // shallow copy of the current object
    copy_on_gpu.data_ = this->data_;
    copy_on_gpu.ndim_ = this->ndim_;
    copy_on_gpu.device_id_ = this->device_id_;
    // copy temporary object to GPU
    hipMemcpy(gpu_ptr, &copy_on_gpu, sizeof(Parcel), hipMemcpyHostToDevice);
    // copy shape and strides data
    this->shape_.copy_to_gpu(&(gpu_ptr->shape_), shape_strides_ptr);
    this->strides_.copy_to_gpu(&(gpu_ptr->strides_), shape_strides_ptr+this->ndim_);
    // nullify data pointer to avoid free data
    copy_on_gpu.data_ = NULL;
    copy_on_gpu.shape_.data() = NULL;
    copy_on_gpu.strides_.data() = NULL;
}

// Free old data
void Parcel::free_current_data(void) {
    // save current device and set device to the corresponding GPU
    int current_device = get_current_device();
    hipSetDevice(this->device_id_);
    // free data
    if (this->data_ != NULL) {
        hipFree(this->data_);
        this->data_ = NULL;
    }
    // finalize: set back the original GPU and unlock the mutex
    hipSetDevice(current_device);
}

// Destructor
Parcel::~Parcel(void) {
    this->free_current_data();
}

}  // namespace merlin
