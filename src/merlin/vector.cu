// Copyright 2024 quocdang1998
#include "merlin/vector.hpp"

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Vector
// ---------------------------------------------------------------------------------------------------------------------

// CUDA copy functions (CPU to GPU)
void vector_cpy_to_gpu(void * dest, const void * src, std::uint64_t size, std::uintptr_t stream_ptr) {
    ::hipMemcpyAsync(dest, src, size, ::hipMemcpyHostToDevice, reinterpret_cast<::hipStream_t>(stream_ptr));
}

// CUDA copy functions (GPU to CPU)
void vector_cpy_from_gpu(void * dest, const void * src, std::uint64_t size, std::uintptr_t stream_ptr) {
    ::hipMemcpyAsync(dest, src, size, ::hipMemcpyDeviceToHost, reinterpret_cast<::hipStream_t>(stream_ptr));
}

}  // namespace merlin
