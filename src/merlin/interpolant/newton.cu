#include "hip/hip_runtime.h"
// Copyright 2022 quocdang1998
#include "merlin/interpolant/newton.hpp"

#include <functional>  // std::bind, std::placeholders
#include <utility> // std::move

#include "merlin/array/copy.hpp"  // merlin::array::array_copy
#include "merlin/array/parcel.hpp"  // merlin::array::Parcel
#include "merlin/array/slice.hpp" // merlin::array::Slice
#include "merlin/cuda/memory.hpp"  // merlin::cuda::Memory
#include "merlin/env.hpp"  // merlin::Environment
#include "merlin/utils.hpp"  // merlin::prod_elements
#include "merlin/interpolant/cartesian_grid.hpp"  // merlin::interpolant::CartesianGrid

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Utils
// --------------------------------------------------------------------------------------------------------------------

// Calculate parallelized divide difference on GPU
void divide_difference_gpu_parallel(const array::Parcel & a1, const array::Parcel & a2, double x1, double x2,
                                    array::Parcel & result, const cuda::Stream & stream) {
    // check for validity
    stream.check_cuda_context();
    // copy data to GPU
    cuda::Memory mem(stream.get_stream_ptr(), a1, a2, result);
    array::Parcel * ptr_a1_on_gpu = const_cast<array::Parcel *>(mem.get<0>());
    array::Parcel * ptr_a2_on_gpu = const_cast<array::Parcel *>(mem.get<1>());
    array::Parcel * ptr_result_on_gpu = const_cast<array::Parcel *>(mem.get<2>());
    std::uint64_t total_malloc_size = mem.get_total_malloc_size();
    // call divide difference algorithm on GPU
    std::uint64_t size = a1.size();
    interpolant::call_divdiff_kernel(ptr_a1_on_gpu, ptr_a2_on_gpu, x1, x2, ptr_result_on_gpu, size,
                                     total_malloc_size, stream.get_stream_ptr());
}

// Calculate coefficients for cartesian grid (supposed shape value == shape of coeff)
void calc_newton_coeffs_gpu_recursive(const interpolant::CartesianGrid & grid, array::Parcel & coeff,
                                      std::uint64_t max_dimension, merlin::Vector<array::Parcel> & sub_slices,
                                      std::uint64_t start_index, const cuda::Stream & stream) {
    // get associated 1D grid to calculate on
    std::uint64_t ndim = grid.ndim();
    std::uint64_t current_dim = ndim - coeff.ndim();
    if (current_dim > max_dimension) {
        return;
    }
    const Vector<double> & grid_vector = grid.grid_vectors()[current_dim];
    // trivial case (1D)
    if (coeff.ndim() == 1) {
        for (std::uint64_t i = 1; i < coeff.shape()[0]; i++) {
            for (std::uint64_t k = coeff.shape()[0]-1; k >=i; k--) {
                long double coeff_calc = (coeff.get({k}) - coeff.get({k-1})) / (grid_vector[k] - grid_vector[k-i]);
                coeff.set({k}, coeff_calc);
            }
        }
        return;
    }
    // calculate divdiff on dim i-th
    for (std::uint64_t i = 1; i < coeff.shape()[0]; i++) {
        for (std::uint64_t k = coeff.shape()[0]-1; k >= i; k--) {
            // get NdData of sub slice
            Vector<array::Slice> slice_k(coeff.ndim()), slice_k_1(coeff.ndim());
            slice_k[0] = array::Slice({k});
            slice_k_1[0] = array::Slice({k-1});
            const array::Parcel array_k(coeff, slice_k);
            const array::Parcel array_k_1(coeff, slice_k_1);
            array::Parcel array_result(coeff, slice_k);
            // calculate divide difference
            divide_difference_gpu_parallel(array_k, array_k_1, grid_vector[k], grid_vector[k-i], array_result, stream);
        }
    }
    stream.synchronize();
    // calculate new start index jump
    intvec shape_other_dims;
    intvec total_dim = grid.get_grid_shape();
    shape_other_dims.assign(total_dim.begin()+current_dim+1, total_dim.begin()+max_dimension+1);
    std::uint64_t start_index_jump = prod_elements(shape_other_dims);
    // recursively calculate divide difference for dimension from i-1-th
    for (std::uint64_t i = 0; i < coeff.shape()[0]; i++) {
        // calculate new start index
        std::uint64_t new_start_index = start_index + i*start_index_jump;
        // get array assigned to slice
        Vector<array::Slice> slice_i(coeff.ndim());
        slice_i[0] = array::Slice({static_cast<std::uint64_t>(i)});
        array::Parcel array_coeff_i(coeff, slice_i);
        array_coeff_i.remove_dim(0);
        calc_newton_coeffs_gpu_recursive(grid, array_coeff_i, max_dimension, sub_slices, new_start_index, stream);
        // push instance to vector
        if (current_dim == max_dimension) {
            sub_slices[new_start_index] = array::Parcel(coeff, slice_i);
        }
    }
}





// Calculate Lagrange interpolation coefficients on a full Cartesian grid using GPU
void interpolant::calc_newton_coeffs_gpu(const interpolant::CartesianGrid & grid, const array::Parcel & value,
                                         array::Parcel & coeff, const cuda::Stream & stream) {
    // check for validity
    stream.check_cuda_context();
    // get associated 1D grid to calculate on
    std::uint64_t ndim = grid.ndim();
    // copy value to coeff
    if (&coeff != &value) {
        auto copy_func = std::bind(::hipMemcpyPeer, std::placeholders::_1, coeff.device().id(),
                                   std::placeholders::_2, value.device().id(), std::placeholders::_3);
        array::array_copy(&coeff, &value, copy_func);
    }
    // get max recursive dimension
    static std::uint64_t parallel_limit = Environment::parallel_chunk;
    intvec total_shape = grid.get_grid_shape();
    std::uint64_t cumulative_size = 1, dim_max = 0;
    while (dim_max < ndim) {
        cumulative_size *= total_shape[dim_max];
        if (cumulative_size >= parallel_limit) {
            break;
        }
        dim_max++;
    }
    // trivial case: size too small
    if (dim_max == ndim) {
        interpolant::calc_newton_coeffs_single_core(grid, coeff);
        return;
    }
    // recursive calculation
    Vector<array::Parcel> sub_slices = make_vector<array::Parcel>(cumulative_size);
    calc_newton_coeffs_gpu_recursive(grid, coeff, dim_max, sub_slices, 0, stream);
    // parallel calculation after that
    for (std::int64_t i = 0; i < sub_slices.size(); i++) {
        // interpolant::calc_newton_coeffs_single_core(grid, sub_slices[i]);
    }
}

}  // namespace merlin
