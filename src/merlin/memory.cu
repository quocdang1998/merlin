// Copyright 2024 quocdang1998
#include "merlin/memory.hpp"

#include "merlin/logger.hpp"  // merlin::Fatal, merlin::Warning

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Utility
// ---------------------------------------------------------------------------------------------------------------------

// Throw CUDA error
static inline void check_cuda_error(::hipError_t error, const std::string & step_name) {
    if (error != 0) {
        Fatal<cuda_runtime_error>("{} failed with error \"{}\".\n", step_name, ::hipGetErrorString(error));
    }
}

// ---------------------------------------------------------------------------------------------------------------------
// CPU Memory Allocator
// ---------------------------------------------------------------------------------------------------------------------

// Allocate page-locked CPU memory
void * mem_alloc_host(std::size_t size) {
    void * allocated_mem;
    check_cuda_error(::hipHostMalloc(&allocated_mem, size), "Malloc page-locked memory");
    return allocated_mem;
}

// Free page-locked CPU memory
void mem_free_host(void * ptr) { check_cuda_error(::hipHostFree(ptr), "Free page-locked memory"); }

// Pin a pre-allocated CPU memory
bool mem_register_host(void * ptr, std::size_t size) {
    ::hipError_t error = ::hipHostRegister(ptr, size, 0);
    if (error == ::hipErrorAlreadyMapped) {
        Warning("The memory has already been mapped. No registration of memory pages was perfermed.\n");
        return false;
    }
    check_cuda_error(error, "Register page-locked memory");
    return true;
}

// Unpin a pre-allocated CPU memory
bool mem_unregister_host(void * ptr) {
    ::hipError_t error = ::hipHostUnregister(ptr);
    if (error == ::hipErrorNotMapped) {
        Warning("The memory has not been mapped. No unregistration operation was performed.\n");
        return false;
    }
    check_cuda_error(error, "Unregister page-locked memory");
    return true;
}

// ---------------------------------------------------------------------------------------------------------------------
// GPU Memory Allocator
// ---------------------------------------------------------------------------------------------------------------------

// Asynchronously allocate memory on the current GPU
void mem_alloc_device(void ** ptr, std::size_t size, std::uintptr_t stream_ptr) {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    check_cuda_error(::hipMallocAsync(ptr, size, hip_stream), "Asynchronous memory allocation on GPU");
}

// Asynchronously deallocate memory on the current GPU
void mem_free_device(void * ptr, std::uintptr_t stream_ptr) {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    check_cuda_error(::hipFreeAsync(ptr, hip_stream), "Asynchronous memory deallocation on GPU");
}

// Asynchronously deallocate memory on the current GPU (without exception)
void mem_free_device_noexcept(void * ptr, std::uintptr_t stream_ptr) noexcept {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    ::hipFreeAsync(ptr, hip_stream);
}

// ---------------------------------------------------------------------------------------------------------------------
// CPU-GPUs Data Transfer
// ---------------------------------------------------------------------------------------------------------------------

// Asynchronously copy data from CPU to GPU
void memcpy_cpu_to_gpu(void * dest, const void * src, std::size_t size, std::uintptr_t stream_ptr) {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    check_cuda_error(::hipMemcpyAsync(dest, src, size, ::hipMemcpyHostToDevice, hip_stream),
                     "Asynchronous memcpy from CPU to GPU");
}

// Asynchronously copy data from GPU to CPU
void memcpy_gpu_to_cpu(void * dest, const void * src, std::size_t size, std::uintptr_t stream_ptr) {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    check_cuda_error(::hipMemcpyAsync(dest, src, size, ::hipMemcpyDeviceToHost, hip_stream),
                     "Asynchronous memcpy from GPU to CPU");
}

// Asynchronously copy data between two locations on the global memory of a GPU
void memcpy_gpu(void * dest, const void * src, std::size_t size, std::uintptr_t stream_ptr) {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    check_cuda_error(::hipMemcpyAsync(dest, src, size, ::hipMemcpyDeviceToDevice, hip_stream),
                     "Asynchronous memcpy within a GPU");
}

// Asynchronously copy data between two GPUs
void memcpy_peer_gpu(void * dest, const void * src, std::size_t size, int dest_gpu, int src_gpu,
                     std::uintptr_t stream_ptr) {
    ::hipStream_t hip_stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    check_cuda_error(::hipMemcpyPeerAsync(dest, dest_gpu, src, src_gpu, size, hip_stream),
                     "Asynchronous memcpy between two GPUs");
}

}  // namespace merlin
