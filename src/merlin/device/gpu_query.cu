#include "hip/hip_runtime.h"
// Copyright 2022 quocdang1998
#include "merlin/device/gpu_query.hpp"

#include <cstdio>  // std::printf
#include <map>  // std::map

#include "merlin/logger.hpp"  // FAILURE, cuda_runtime_error

namespace merlin {

// Get total number of GPU
int get_device_count(void) {
    int count;
    hipGetDeviceCount(&count);
    return count;
}

// Convert GPU major.minor version to number of CUDA core
// Adapted from function _ConvertSMVer2Cores, see https://github.com/NVIDIA/cuda-samples/blob/master/Common/hip/hip_runtime_api.h
static int convert_SM_version_to_core(int major, int minor) {
    std::map<int, int> num_gpu_arch_cores_per_SM = {
        {0x30, 192},
        {0x32, 192},
        {0x35, 192},
        {0x37, 192},
        {0x50, 128},
        {0x52, 128},
        {0x53, 128},
        {0x60,  64},
        {0x61, 128},
        {0x62, 128},
        {0x70,  64},
        {0x72,  64},
        {0x75,  64},
        {0x80,  64},
        {0x86, 128},
        {0x87, 128}
    };
    int SM = (major << 4) + minor;
    if (num_gpu_arch_cores_per_SM.find(SM) == num_gpu_arch_cores_per_SM.end()) {
        FAILURE(cuda_runtime_error, "Cannot detect SM number in the map \"num_gpu_arch_cores_per_SM\".\n");
    }
    return num_gpu_arch_cores_per_SM[SM];
}

// Get limit of a given device
static void print_limit_of_one_device(int device) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    // Device name
    std::printf("    Name : %s.\n", prop.name);
    // Max multi-processor
    std::printf("    Number of multiprocessors on the device: %d.\n", prop.multiProcessorCount);
    // Number of CUDA core
    int core_per_multiprocessor = convert_SM_version_to_core(prop.major, prop.minor);
    std::printf("    Number of CUDA core per multiprocessor: %d.\n", core_per_multiprocessor);
    std::printf("    Total number of CUDA core: %d.\n", core_per_multiprocessor*prop.multiProcessorCount);
    // Max thread per multi-processor
    std::printf("    Maximum resident threads per multiprocessor: %d.\n", prop.maxThreadsPerMultiProcessor);

    // Max threads per block
    std::printf("    Maximum number of threads per block: %d.\n", prop.maxThreadsPerBlock);
    // Max blockDim
    std::printf("    Maximum (x,y,z)-dimension of a block: (%d, %d, %d).\n",
                prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    // Max gridDim
    std::printf("    Maximum (x,y,z)-dimension of a grid: (%d, %d, %d).\n",
                prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    // Total global memory
    std::printf("    Total amount of global memory: %f GB.\n", static_cast<float>(prop.totalGlobalMem)/1073741824.0f);
    // Max shared memory per block
    std::printf("    Maximum amount of shared memory available to a thread block: %zu bytes.\n", prop.sharedMemPerBlock);
    // Max constant memory
    std::printf("    Memory available on device for __constant__ variables in a CUDA C kernel: %zu bytes.\n",
                prop.totalConstMem);
}

// Print limit of a device
void print_device_limit(int device) {
    int tot_device = get_device_count();
    if (device != -1) {
        // Check valid argument
        if ((device < 0) || (device >= tot_device)) {
            FAILURE(std::invalid_argument, "Invalid number of device (expected value >= 0 and < %d, got %d.\n",
                    tot_device, device);
        }
        // Print only one device
        std::printf("Chosen device: %d.\n", device);
        print_limit_of_one_device(device);
        return;
    }
    // if no device is selected, query for all devices
    for (int i = 0; i < tot_device; i++) {
        std::printf("GPU Id: %d.\n", i);
        print_limit_of_one_device(i);
    }
}

// Add 2 integer on GPU
__global__ static void add_2_int_on_gpu(int * p_a, int * p_b, int * p_result) {
    *p_result = *p_a + *p_b;
}

// Test functionality of one GPU
static bool test_gpu_on_a_device(int device) {
    // initialize
    int cpu_int[3] = {2, 4, 0};
    int * gpu_int;
    hipError_t err_;
    int reference = cpu_int[0] + cpu_int[1];
    // set device
    err_ = hipSetDevice(device);
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipSetDevice for id = %d failed with message \"%s\".\n",
                device, hipGetErrorName(err_));
    }
    // malloc
    err_ = hipMalloc(&gpu_int, 3*sizeof(int));
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipMalloc failed with message \"%s\".\n", hipGetErrorName(err_));
    }
    // copy to gpu
    err_ = hipMemcpy(gpu_int, cpu_int, 3*sizeof(int), hipMemcpyHostToDevice);
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipMemcpyHostToDevice failed with message \"%s\".\n", hipGetErrorName(err_));
    }
    // launch kernel
    add_2_int_on_gpu<<<1, 1>>>(gpu_int, gpu_int+1, gpu_int+2);
    hipDeviceSynchronize();
    err_ = hipGetLastError();
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Launch kernel failed with message \"%s\".\n", hipGetErrorName(err_));
    }
    // copy to cpu
    err_ = hipMemcpy(cpu_int, gpu_int, 3*sizeof(int), hipMemcpyDeviceToHost);
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipMemcpyDeviceToHost failed with message \"%s\".\n", hipGetErrorName(err_));
    }
    // check result
    if (cpu_int[2] != reference) {
        WARNING("Expected result of adding %d and %d on GPU ID %d is %d, got %d.\n",
                cpu_int[0], cpu_int[1], device, reference, cpu_int[2]);
        return false;
    }
    return true;
}

// Test functionality of GPU
bool test_gpu(int device) {
    int tot_device = get_device_count();
    bool result = true;
    if (device != -1) {
        // Check valid argument
        if ((device < 0) || (device >= tot_device)) {
            FAILURE(std::invalid_argument, "Invalid number of device (expected value >= 0 and < %d, got %d.\n",
                    tot_device, device);
        }
        // Test for only one device
        std::printf("Checking device: %d...", device);
        result = result && test_gpu_on_a_device(device);
        if (!result) {
            WARNING("\rCheck on device %d has failed.\n", device);
        } else {
            std::printf("\r");
        }
        return result;
    }
    // if no device is selected, test for all devices
    for (int i = 0; i < tot_device; i++) {
        std::printf("Checking device: %d...", i);
        result = result && test_gpu_on_a_device(i);
        if (!result) {
            WARNING("\rCheck on device %d has failed.\n", i);
        } else {
            std::printf("\r");
        }
    }
    return result;
}

}  // namespace merlin
