// Copyright 2022 quocdang1998
#include "merlin/device/context.hpp"

#include "hip/hip_runtime.h"  // hipCtxCreate, hipCtxDestroy, hipCtx_t

namespace merlin::device {

// -------------------------------------------------------------------------------------------------------------------------
// Context
// -------------------------------------------------------------------------------------------------------------------------

// Member constructor
Context::Context(const Device & gpu, Context::Flags flag) {
    hipCtx_t ctx;
    hipError_t err_ = hipCtxCreate(&ctx, static_cast<unsigned int>(flag), gpu.id());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create context failed with message \"%s\".\n", cuda_get_error_name(err_));
    }
    this->context_ = reinterpret_cast<std::uintptr_t>(ctx);
    this->device_ = gpu;
}

// Push the context to the stack
void Context::push_current(void) {
    if (this->attached_) {
        FAILURE(cuda_runtime_error, "The current context is being attached to the CPU process\n");
    }
    hipCtx_t ctx = reinterpret_cast<hipCtx_t>(this->context_);
    hipError_t err_ = hipCtxPushCurrent(ctx);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Push context to current failed with message \"%s\".\n", cuda_get_error_name(err_));
    }
    this->attached_ = false;
}

// Pop the context out of the stack
Context & Context::pop_current(void) {
    if (!(this->attached_)) {
        FAILURE(cuda_runtime_error, "The current context is not being attached to any processes\n");
    }
    hipCtx_t ctx = reinterpret_cast<hipCtx_t>(this->context_);
    hipError_t err_ = hipCtxPopCurrent(&ctx);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Pop current context failed with message \"%s\".\n", cuda_get_error_name(err_));
    }
    this->attached_ = true;
    return *this;
}

// Check if the context is the top of context stack
bool Context::is_current(void) {
    hipCtx_t current_ctx;
    hipError_t err_ = hipCtxGetCurrent(&current_ctx);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", cuda_get_error_name(err_));
    }
    return (this->context_ == reinterpret_cast<std::uintptr_t>(current_ctx));
}

// Set current context at the top of the stack
void Context::set_current(void) {
    if (!(this->attached_)) {
        FAILURE(cuda_runtime_error, "The current context is not being attached to any process\n");
    }
    hipCtx_t current_ctx = reinterpret_cast<hipCtx_t>(this->context_);
    hipError_t err_ = hipCtxSetCurrent(current_ctx);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Set current context failed with message \"%s\".\n", cuda_get_error_name(err_));
    }
}

// Create list of primary contexts
void Context::create_primary_context_list(void) {
    int num_gpu = Device::get_num_gpu();
    // skip when the primary contexts are initialized
    if (num_gpu == Context::primary_contexts.size()) {
        return;
    }
    for (int i = 0; i < num_gpu; i++) {
        Context::primary_contexts.emplace_back(Context::create_primary_context(Device(i)));
    }
}

// Create primary context instance assigned to a GPU
Context Context::create_primary_context(const Device & gpu) {
    Context result;
    result.device_ = gpu;
    auto [active, _] = result.get_primary_ctx_state(gpu);
    result.attached_ = active;
    return result;
}

// Get state of the primary context
std::pair<bool, Context::Flags> Context::get_primary_ctx_state(const Device & gpu) {
    unsigned int flags;
    int active;
    hipError_t err_ = hipDevicePrimaryCtxGetState(gpu.id(), &flags, &active);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get state of primary context of GPU %d failed with message \"%s\".\n",
                gpu.id(), cuda_get_error_name(err_));
    }
    return std::pair<bool, Context::Flags>(bool(active), Context::Flags(flags));
}

// Set flag for primary context
void Context::set_flag_primary_context(const Device & gpu, Context::Flags flag) {
    hipError_t err_ = hipDevicePrimaryCtxSetFlags(gpu.id(), static_cast<unsigned int>(flag));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Set flag to primary context of GPU %d failed with message \"%s\".\n",
                gpu.id(), cuda_get_error_name(err_));
    }
}

// Destructor
Context::~Context(void) {
    if (this->context_ != 0) {
        hipCtxDestroy(reinterpret_cast<hipCtx_t>(this->context_));
    }
}

}  // namespace merlin::device
