// Copyright 2023 quocdang1998
#include "merlin/candy/optimizer.hpp"

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Optimizer
// ---------------------------------------------------------------------------------------------------------------------

// Copy the optimizer from CPU to a pre-allocated memory on GPU
void * candy::Optimizer::copy_to_gpu(candy::Optimizer * gpu_ptr, void * dynamic_data_ptr,
                                     std::uintptr_t stream_ptr) const {
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    // create an instant similar to the copy on GPU
    candy::Optimizer copy_on_gpu;
    copy_on_gpu.static_data = this->static_data;
    copy_on_gpu.dynamic_data = reinterpret_cast<double *>(dynamic_data_ptr);
    copy_on_gpu.dynamic_size = this->dynamic_size;
    // copy the clone and dynamic data to GPU
    ::hipMemcpyAsync(gpu_ptr, &copy_on_gpu, sizeof(candy::Optimizer), ::hipMemcpyHostToDevice, stream);
    ::hipMemcpyAsync(dynamic_data_ptr, this->dynamic_data, sizeof(double) * this->dynamic_size,
                      ::hipMemcpyHostToDevice, stream);
    // nullify pointer on the clone
    double * returned_ptr = copy_on_gpu.dynamic_data;
    copy_on_gpu.dynamic_data = nullptr;
    return returned_ptr + this->dynamic_size;
}

// Copy data from GPU to CPU
void * candy::Optimizer::copy_from_gpu(double * data_from_gpu, std::uintptr_t stream_ptr) noexcept {
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    ::hipMemcpyAsync(this->dynamic_data, data_from_gpu, sizeof(double) * this->dynamic_size, ::hipMemcpyDeviceToHost,
                      stream);
    return reinterpret_cast<void *>(data_from_gpu + this->dynamic_size);
}

}  // namespace merlin
