// Copyright 2023 quocdang1998
#include "merlin/candy/model.hpp"

#include <cstdint>  // std::uintptr_t

#include "merlin/utils.hpp"  // merlin::ptr_to_subsequence

namespace merlin {

// Copy data to a pre-allocated memory
void * candy::Model::copy_to_gpu(candy::Model * gpu_ptr, void * parameters_data_ptr, std::uintptr_t stream_ptr) const {
    // initialize buffer to store data of the copy before cloning it to GPU
    candy::Model copy_on_gpu;
    // shallow copy of parameters, rshape and param vectors
    double * parameters_ptr = reinterpret_cast<double *>(parameters_data_ptr);
    copy_on_gpu.parameters_.data() = parameters_ptr;
    copy_on_gpu.parameters_.size() = this->num_params();
    std::uint64_t * rshape_ptr = reinterpret_cast<std::uint64_t *>(parameters_ptr + this->num_params());
    copy_on_gpu.rshape_.data() = rshape_ptr;
    copy_on_gpu.rshape_.size() = this->ndim();
    Vector<double *> gpu_param_vector = ptr_to_subsequence(parameters_ptr, this->rshape_);
    double ** param_vectors_ptr = reinterpret_cast<double **>(rshape_ptr + this->ndim());
    copy_on_gpu.param_vectors_.data() = param_vectors_ptr;
    copy_on_gpu.param_vectors_.size() = this->ndim();
    copy_on_gpu.rank_ = this->rank_;
    // copy data of each vector
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    ::hipMemcpyAsync(parameters_ptr, this->parameters_.data(), this->num_params() * sizeof(double),
                      ::hipMemcpyHostToDevice, stream);
    ::hipMemcpyAsync(rshape_ptr, this->rshape_.data(), this->ndim() * sizeof(std::uint64_t), ::hipMemcpyHostToDevice,
                      stream);
    ::hipMemcpyAsync(param_vectors_ptr, gpu_param_vector.data(), this->ndim() * sizeof(double *),
                      ::hipMemcpyHostToDevice, stream);
    // copy temporary object to GPU
    ::hipMemcpyAsync(gpu_ptr, &copy_on_gpu, sizeof(candy::Model), ::hipMemcpyHostToDevice, stream);
    // nullify pointer of temporary object to avoid de-allocate GPU pointer
    copy_on_gpu.parameters_.data() = nullptr;
    copy_on_gpu.rshape_.data() = nullptr;
    copy_on_gpu.param_vectors_.data() = nullptr;
    return reinterpret_cast<void *>(param_vectors_ptr + this->ndim());
}

// Copy data from GPU to CPU
void * candy::Model::copy_from_gpu(double * data_from_gpu, std::uintptr_t stream_ptr) noexcept {
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    ::hipMemcpyAsync(this->parameters_.data(), data_from_gpu, this->num_params() * sizeof(double),
                      ::hipMemcpyDeviceToHost, stream);
    return reinterpret_cast<void *>(data_from_gpu + this->num_params());
}

}  // namespace merlin
