// Copyright 2022 quocdang1998
#include "merlin/array/array.hpp"

#include <cinttypes>   // PRIu64
#include <functional>  // std::bind, std::placeholders

#include "merlin/array/operation.hpp"  // merlin::array::copy
#include "merlin/array/parcel.hpp"     // merlin::array::Parcel
#include "merlin/cuda/device.hpp"      // merlin::cuda::Device
#include "merlin/env.hpp"              // merlin::Environment
#include "merlin/logger.hpp"           // merlin::Fatal, merlin::Warning

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Memory lock (allocated array always stays in the RAM)
// ---------------------------------------------------------------------------------------------------------------------

// Allocate non pageable memory
double * array::allocate_memory(std::uint64_t size) {
    double * result = nullptr;
    ::hipError_t err_ = ::hipHostMalloc(&result, sizeof(double) * size);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Allocate non-pageable memory failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return result;
}

// Pin memory to RAM
void array::cuda_pin_memory(double * ptr, std::uint64_t mem_size) {
    ::hipError_t err_ = ::hipHostRegister(ptr, mem_size, hipHostRegisterDefault);
    if (err_ != 0) {
        Warning("Pin pageable memory failed with message \"") << ::hipGetErrorString(err_) << "\".\n";
    }
}

// Unpin memory
void array::cuda_unpin_memory(double * ptr) {
    ::hipError_t err_ = ::hipHostUnregister(ptr);
    if (err_ != 0) {
        Warning("Unpin paged memory failed with message \"") << ::hipGetErrorString(err_) << "\".\n";
    }
}

// Free non pageable memory
void array::free_memory(double * ptr) {
    ::hipError_t err_ = ::hipHostFree(ptr);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Free non-pageable memory failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
}

// ---------------------------------------------------------------------------------------------------------------------
// Array
// ---------------------------------------------------------------------------------------------------------------------

// Copy data from GPU array
void array::Array::clone_data_from_gpu(const array::Parcel & src, const cuda::Stream & stream) {
    // check GPU of stream
    if (src.device() != stream.get_gpu()) {
        Fatal<cuda_runtime_error>("Cannot copy from GPU array (%d) with stream pointing to another GPU (%d).\n",
                                  src.device(), stream.get_gpu());
    }
    // cast stream
    ::hipStream_t copy_stream = reinterpret_cast<::hipStream_t>(stream.get_stream_ptr());
    // create copy function
    auto copy_func = std::bind(::hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               ::hipMemcpyDeviceToHost, copy_stream);
    // copy data to GPU
    std::uintptr_t current_ctx = src.device().push_context();
    array::copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
    cuda::Device::pop_context(current_ctx);
}

}  // namespace merlin
