// Copyright 2022 quocdang1998
#include "merlin/array/array.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array/parcel.hpp"  // merlin::Parcel
#include "merlin/logger.hpp"  // FAILURE
#include "merlin/array/utils.hpp"  // merlin::array_copy

namespace merlin {

void Array::sync_from_gpu(const Parcel & gpu_array, std::uintptr_t stream) {
    // check device
    int check_result = gpu_array.check_device();
    if (check_result != 0) {
        FAILURE(cuda_runtime_error, "Current GPU is not corresponding (expected ID %d, got ID %d).\n",
                gpu_array.device_id(), gpu_array.device_id() - check_result);
    }
    // cast stream
    hipStream_t copy_stream = reinterpret_cast<hipStream_t>(stream);
    // create copy function
    auto copy_func = std::bind(hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               hipMemcpyDeviceToHost, copy_stream);
    // copy data to GPU
    array_copy(dynamic_cast<NdData *>(this), dynamic_cast<const NdData *>(&gpu_array), copy_func);
}

}  // namespace merlin
