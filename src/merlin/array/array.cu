// Copyright 2022 quocdang1998
#include "merlin/array/array.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array/copy.hpp"  // merlin::array::array_copy
#include "merlin/array/parcel.hpp"  // merlin::array::Parcel
#include "merlin/cuda/device.hpp"  // merlin::cuda::Device
#include "merlin/logger.hpp"  // FAILURE

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Array
// --------------------------------------------------------------------------------------------------------------------

// Copy data from GPU array
void array::Array::clone_data_from_gpu(const array::Parcel & src, const cuda::Stream & stream) {
    // save current gpu
    cuda::Device current_gpu = cuda::Device::get_current_gpu();
    // check GPU of stream
    if (src.device() != stream.get_gpu()) {
        FAILURE(cuda_runtime_error, "Cannot copy from GPU array (%d) with stream pointing to another GPU (%d).\n",
                src.device(), stream.get_gpu());
    }
    // cast stream
    ::hipStream_t copy_stream = reinterpret_cast<::hipStream_t>(stream.get_stream_ptr());
    // create copy function
    auto copy_func = std::bind(::hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               ::hipMemcpyDeviceToHost, copy_stream);
    // copy data to GPU
    src.device().set_as_current();
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
    current_gpu.set_as_current();
}

}  // namespace merlin
