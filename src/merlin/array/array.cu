// Copyright 2022 quocdang1998
#include "merlin/array/array.hpp"

#include <cinttypes>   // PRIu64
#include <functional>  // std::bind, std::placeholders

#include "merlin/array/operation.hpp"  // merlin::array::array_copy
#include "merlin/array/parcel.hpp"     // merlin::array::Parcel
#include "merlin/cuda/device.hpp"      // merlin::cuda::Device
#include "merlin/logger.hpp"           // FAILURE

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Memory lock (allocated array always stays in the RAM)
// ---------------------------------------------------------------------------------------------------------------------

// Allocate non pageable memory
double * array::allocate_memory(std::uint64_t size) {
    double * result = nullptr;
    ::hipError_t err_ = ::hipHostMalloc(&result, sizeof(double) * size);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Allocate non-pageable memory failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return result;
}

// Pin memory to RAM
void array::cuda_pin_memory(double * ptr, std::uint64_t n_elem) {
    ::hipError_t err_ = ::hipHostRegister(ptr, sizeof(double) * n_elem, hipHostRegisterDefault);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Pin pageable memory failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// Free non pageable memory
void array::free_memory(double * ptr) {
    ::hipError_t err_ = ::hipHostFree(ptr);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Free non-pageable memory failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
}

// ---------------------------------------------------------------------------------------------------------------------
// Array
// ---------------------------------------------------------------------------------------------------------------------

// Copy data from GPU array
void array::Array::clone_data_from_gpu(const array::Parcel & src, const cuda::Stream & stream) {
    // save current gpu
    cuda::Device current_gpu = cuda::Device::get_current_gpu();
    // check GPU of stream
    if (src.device() != stream.get_gpu()) {
        FAILURE(cuda_runtime_error, "Cannot copy from GPU array (%d) with stream pointing to another GPU (%d).\n",
                src.device(), stream.get_gpu());
    }
    // cast stream
    ::hipStream_t copy_stream = reinterpret_cast<::hipStream_t>(stream.get_stream_ptr());
    // create copy function
    auto copy_func = std::bind(::hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               ::hipMemcpyDeviceToHost, copy_stream);
    // copy data to GPU
    src.device().set_as_current();
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
    current_gpu.set_as_current();
}

}  // namespace merlin
