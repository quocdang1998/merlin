// Copyright 2022 quocdang1998
#include "merlin/array/array.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array/copy.hpp"  // merlin::array::array_copy
#include "merlin/array/parcel.hpp"  // merlin::array::Parcel
#include "merlin/device/gpu_query.hpp"  // merlin::device::Device
#include "merlin/logger.hpp"  // FAILURE

namespace merlin::array {

// Copy data from GPU array
void Array::sync_from_gpu(const Parcel & gpu_array, std::uintptr_t stream) {
    // check device
    device::Device current_gpu = device::Device::get_current_gpu();
    if (current_gpu != gpu_array.device()) {
        FAILURE(cuda_runtime_error, "Current GPU is not corresponding (expected ID %d, got ID %d).\n",
                gpu_array.device().id(), current_gpu.id());
    }
    // cast stream
    hipStream_t copy_stream = reinterpret_cast<hipStream_t>(stream);
    // create copy function
    auto copy_func = std::bind(hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               hipMemcpyDeviceToHost, copy_stream);
    // copy data to GPU
    array_copy(dynamic_cast<NdData *>(this), dynamic_cast<const NdData *>(&gpu_array), copy_func);
}

}  // namespace merlin::array
