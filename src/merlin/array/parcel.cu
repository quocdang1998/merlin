// Copyright 2022 quocdang1998
#include "merlin/array/parcel.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array/array.hpp"  // merlin::array::Array
#include "merlin/array/copy.hpp"  // merlin::array::contiguous_strides, merlin::array::array_copy
#include "merlin/array/slice.hpp"  // merlin::array::Slice
#include "merlin/logger.hpp"  // FAILURE
#include "merlin/utils.hpp"  // merlin::contiguous_to_ndim_idx, merlin::inner_prod

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Parcel
// --------------------------------------------------------------------------------------------------------------------

// Free old data
void array::Parcel::free_current_data(void) {
    // lock mutex
    array::Parcel::mutex_.lock();
    // switch to appropriate context
    this->context_.push_current();
    // free data
    if ((this->data_ != nullptr) && this->release_) {
        ::hipFree(this->data_);
        this->data_ = nullptr;
    }
    // finalize: set back the original GPU and unlock the mutex
    this->context_.pop_current();
    array::Parcel::mutex_.unlock();
}

// Constructor from shape vector
array::Parcel::Parcel(const intvec & shape) : array::NdData(shape) {
    // allocate data
    this->release_ = true;
    ::hipError_t err_ = ::hipMalloc(&(this->data_), sizeof(double) * this->size());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    // set device and context
    this->device_ = cuda::Device::get_current_gpu();
    this->context_ = cuda::Context::get_current();
}

// Copy constructor
array::Parcel::Parcel(const array::Parcel & src) : array::NdData(src) {
    // get device id and context
    this->device_ = cuda::Device::get_current_gpu();
    this->context_ = cuda::Context::get_current();
    // reform strides vector
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(double));
    // allocate data
    this->release_ = true;
    ::hipError_t err_ = ::hipMalloc(&(this->data_), sizeof(double) * this->size());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    // create copy function and copy data to GPU
    if (this->device_ != src.device_) {
        auto copy_func = std::bind(::hipMemcpyPeer, std::placeholders::_1, this->device_.id(),
                                   std::placeholders::_2, src.device_.id(), std::placeholders::_3);
        array::array_copy(this, &src, copy_func);
    } else {
        auto copy_func = std::bind(::hipMemcpy, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                                   ::hipMemcpyDeviceToDevice);
        array::array_copy(this, &src, copy_func);
    }
}

// Copy assignement
array::Parcel & array::Parcel::operator=(const array::Parcel & src) {
    // free old data
    this->free_current_data();
    // copy metadata and reform strides vector
    this->array::NdData::operator=(src);
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(double));
    // get device and context
    this->device_ = cuda::Device::get_current_gpu();
    this->context_ = cuda::Context::get_current();
    // allocate data
    this->release_ = true;
    hipError_t err_ = ::hipMalloc(&(this->data_), sizeof(double) * this->size());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // create copy function and copy data to GPU
    if (this->device_ != src.device_) {
        auto copy_func = std::bind(::hipMemcpyPeer, std::placeholders::_1, this->device_.id(),
                                   std::placeholders::_2, src.device_.id(), std::placeholders::_3);
        array::array_copy(this, &src, copy_func);
    } else {
        auto copy_func = std::bind(::hipMemcpy, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                                   ::hipMemcpyDeviceToDevice);
        array::array_copy(this, &src, copy_func);
    }
    return *this;
}

// Move constructor
array::Parcel::Parcel(array::Parcel && src) : array::NdData(src) {
    // move device id and context
    this->device_ = src.device_;
    this->context_ = src.context_;
    // take over pointer to source
    this->release_ = src.release_;
    src.data_ = nullptr;
}

// Move assignment
array::Parcel & array::Parcel::operator=(array::Parcel && src) {
    // free old data
    this->free_current_data();
    // move device id and context
    this->device_ = src.device_;
    this->context_ = src.context_;
    // copy metadata
    this->array::NdData::operator=(src);
    // take over pointer to source
    src.data_ = nullptr;
    this->release_ = src.release_;
    return *this;
}

// Get value of element at a n-dim index
double array::Parcel::get(const intvec & index) const {
    std::uint64_t leap = inner_prod(index, this->strides_);
    std::uintptr_t data_ptr = reinterpret_cast<std::uintptr_t>(this->data_) + leap;
    double result;
    array::Parcel::mutex_.lock();
    bool must_pop_current = false;
    if (!(this->context_.is_current())) {
        this->context_.push_current();
        must_pop_current = true;
    }
    ::hipMemcpy(&result, reinterpret_cast<double *>(data_ptr), sizeof(double), ::hipMemcpyDeviceToHost);
    if (must_pop_current) {
        this->context_.pop_current();
    }
    array::Parcel::mutex_.unlock();
    return result;
}

// Get value of element at a C-contiguous index
double array::Parcel::get(std::uint64_t index) const {
    return this->get(contiguous_to_ndim_idx(index, this->shape()));
}

// Set value of element at a n-dim index
void array::Parcel::set(const intvec index, double value) {
    std::uint64_t leap = inner_prod(index, this->strides_);
    std::uintptr_t data_ptr = reinterpret_cast<std::uintptr_t>(this->data_) + leap;
    array::Parcel::mutex_.lock();
    bool must_pop_current = false;
    if (!this->context_.is_current()) {
        this->context_.push_current();
        must_pop_current = true;
    }
    ::hipMemcpy(reinterpret_cast<double *>(data_ptr), &value, sizeof(double), ::hipMemcpyHostToDevice);
    if (must_pop_current) {
        this->context_.pop_current();
    }
    array::Parcel::mutex_.unlock();
}

// Set value of element at a C-contiguous index
void array::Parcel::set(std::uint64_t index, double value) {
    this->set(contiguous_to_ndim_idx(index, this->shape()), value);
}

// Transfer data to GPU
void array::Parcel::transfer_data_to_gpu(const array::Array & cpu_array, const cuda::Stream & stream) {
    // get device id
    stream.check_cuda_context();
    // cast stream
    ::hipStream_t copy_stream = reinterpret_cast<::hipStream_t>(stream.get_stream_ptr());
    // create copy function
    auto copy_func = std::bind(::hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               ::hipMemcpyHostToDevice, copy_stream);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&cpu_array), copy_func);
}

// Copy data to a pre-allocated memory
void * array::Parcel::copy_to_gpu(array::Parcel * gpu_ptr, void * shape_strides_ptr, std::uintptr_t stream_ptr) const {
    // initialize buffer to store data of the copy before cloning it to GPU
    array::Parcel copy_on_gpu;
    // shallow copy of the current object
    copy_on_gpu.data_ = this->data_;
    copy_on_gpu.ndim_ = this->ndim_;
    copy_on_gpu.device_ = this->device_;
    copy_on_gpu.context_ = this->context_;
    // copy temporary object to GPU
    ::hipMemcpyAsync(gpu_ptr, &copy_on_gpu, sizeof(array::Parcel), ::hipMemcpyHostToDevice,
                      reinterpret_cast<::hipStream_t>(stream_ptr));
    // copy shape and strides data
    void * strides_data_ptr_gpu = this->shape_.copy_to_gpu(&(gpu_ptr->shape_), shape_strides_ptr, stream_ptr);
    void * result_ptr = this->strides_.copy_to_gpu(&(gpu_ptr->strides_), strides_data_ptr_gpu, stream_ptr);
    // nullify data pointer to avoid free data
    copy_on_gpu.data_ = nullptr;
    copy_on_gpu.shape_.data() = nullptr;
    copy_on_gpu.strides_.data() = nullptr;
    return result_ptr;
}

// Destructor
array::Parcel::~Parcel(void) {
    this->free_current_data();
}

}  // namespace merlin
