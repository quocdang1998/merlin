// Copyright 2022 quocdang1998
#include "merlin/array/parcel.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array/array.hpp"  // merlin::array::Array
#include "merlin/array/copy.hpp"  // merlin::array::contiguous_strides, merlin::array::array_copy
#include "merlin/array/slice.hpp"  // merlin::array::Slice
#include "merlin/logger.hpp"  // FAILURE
#include "merlin/utils.hpp"  // merlin::contiguous_to_ndim_idx, merlin::inner_prod

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Parcel
// --------------------------------------------------------------------------------------------------------------------

// Free old data
void array::Parcel::free_current_data(void) {
    // lock mutex
    array::Parcel::mutex_.lock();
    // switch to appropriate context
    cuda::Device current_device = cuda::Device::get_current_gpu();
    this->device_.set_as_current();
    // free data
    if ((this->data_ != nullptr) && this->release_) {
        ::hipFree(this->data_);
        this->data_ = nullptr;
    }
    // finalize: set back the original GPU and unlock the mutex
    current_device.set_as_current();
    array::Parcel::mutex_.unlock();
}

// Constructor from shape vector
array::Parcel::Parcel(const intvec & shape) : array::NdData(shape) {
    // allocate data
    this->release_ = true;
    ::hipError_t err_ = ::hipMalloc(&(this->data_), sizeof(double) * this->size());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
}

// Copy constructor
array::Parcel::Parcel(const array::Parcel & src) : array::NdData(src) {
    // get device id and context
    this->device_ = cuda::Device::get_current_gpu();
    // reform strides vector
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(double));
    // allocate data
    this->release_ = true;
    ::hipError_t err_ = ::hipMalloc(&(this->data_), sizeof(double) * this->size());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    // create copy function
    auto copy_func = std::bind(::hipMemcpyPeer, std::placeholders::_1, this->device_.id(),
                               std::placeholders::_2, src.device_.id(), std::placeholders::_3);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
}

// Copy assignement
array::Parcel & array::Parcel::operator=(const array::Parcel & src) {
    // free old data
    this->free_current_data();
    // copy metadata and reform strides vector
    this->array::NdData::operator=(src);
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(double));
    // get device id
    this->device_ = cuda::Device::get_current_gpu();
    // allocate data
    this->release_ = true;
    hipError_t err_ = ::hipMalloc(&(this->data_), sizeof(double) * this->size());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // create copy function
    auto copy_func = std::bind(::hipMemcpyPeer, std::placeholders::_1, this->device_.id(),
                               std::placeholders::_2, src.device_.id(), std::placeholders::_3);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
    return *this;
}

// Move constructor
array::Parcel::Parcel(array::Parcel && src) : array::NdData(src) {
    // move device id and context
    this->device_ = src.device_;
    // take over pointer to source
    this->release_ = src.release_;
    src.data_ = nullptr;
}

// Move assignment
array::Parcel & array::Parcel::operator=(array::Parcel && src) {
    // free old data
    this->free_current_data();
    // move device id
    this->device_ = src.device_;
    // copy metadata
    this->array::NdData::operator=(src);
    // take over pointer to source
    src.data_ = nullptr;
    this->release_ = src.release_;
    return *this;
}

// Get value of element at a n-dim index
double array::Parcel::get(const intvec & index) const {
    std::uint64_t leap = inner_prod(index, this->strides_);
    std::uintptr_t data_ptr = reinterpret_cast<std::uintptr_t>(this->data_) + leap;
    double result;
    array::Parcel::mutex_.lock();
    cuda::Device current_device = cuda::Device::get_current_gpu();
    this->device_.set_as_current();
    ::hipMemcpy(&result, reinterpret_cast<double *>(data_ptr), sizeof(double), ::hipMemcpyDeviceToHost);
    current_device.set_as_current();
    array::Parcel::mutex_.unlock();
    return result;
}

// Get value of element at a C-contiguous index
double array::Parcel::get(std::uint64_t index) const {
    return this->get(contiguous_to_ndim_idx(index, this->shape()));
}

// Set value of element at a n-dim index
void array::Parcel::set(const intvec index, double value) {
    std::uint64_t leap = inner_prod(index, this->strides_);
    std::uintptr_t data_ptr = reinterpret_cast<std::uintptr_t>(this->data_) + leap;
    array::Parcel::mutex_.lock();
    cuda::Device current_device = cuda::Device::get_current_gpu();
    this->device_.set_as_current();
    ::hipMemcpy(reinterpret_cast<double *>(data_ptr), &value, sizeof(double), ::hipMemcpyHostToDevice);
    current_device.set_as_current();
    array::Parcel::mutex_.unlock();
}

// Set value of element at a C-contiguous index
void array::Parcel::set(std::uint64_t index, double value) {
    this->set(contiguous_to_ndim_idx(index, this->shape()), value);
}

// Transfer data to GPU
void array::Parcel::transfer_data_to_gpu(const array::Array & cpu_array, const cuda::Stream & stream) {
    // get device id
    stream.check_cuda_context();
    this->device_ = stream.get_gpu();
    // cast stream
    ::hipStream_t copy_stream = reinterpret_cast<::hipStream_t>(stream.get_stream_ptr());
    // create copy function
    auto copy_func = std::bind(::hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               ::hipMemcpyHostToDevice, copy_stream);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&cpu_array), copy_func);
}

// Copy data to a pre-allocated memory
void * array::Parcel::copy_to_gpu(array::Parcel * gpu_ptr, void * shape_strides_ptr) const {
    // initialize buffer to store data of the copy before cloning it to GPU
    array::Parcel copy_on_gpu;
    // shallow copy of the current object
    copy_on_gpu.data_ = this->data_;
    copy_on_gpu.ndim_ = this->ndim_;
    copy_on_gpu.device_ = this->device_;
    // copy temporary object to GPU
    ::hipMemcpy(gpu_ptr, &copy_on_gpu, sizeof(array::Parcel), ::hipMemcpyHostToDevice);
    // copy shape and strides data
    void * strides_data_ptr_gpu = this->shape_.copy_to_gpu(&(gpu_ptr->shape_), shape_strides_ptr);
    void * result_ptr = this->strides_.copy_to_gpu(&(gpu_ptr->strides_), strides_data_ptr_gpu);
    // nullify data pointer to avoid free data
    copy_on_gpu.data_ = nullptr;
    copy_on_gpu.shape_.data() = nullptr;
    copy_on_gpu.strides_.data() = nullptr;
    return result_ptr;
}

// Destructor
array::Parcel::~Parcel(void) {
    this->free_current_data();
}

}  // namespace merlin
