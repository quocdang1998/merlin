// Copyright 2022 quocdang1998
#include "merlin/array/parcel.hpp"

#include <functional>  // std::bind, std::placeholders

#include "merlin/array/array.hpp"  // merlin::array::Array
#include "merlin/array/copy.hpp"  // merlin::array::contiguous_strides, merlin::array::array_copy
#include "merlin/logger.hpp"  // FAILURE

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Parcel
// --------------------------------------------------------------------------------------------------------------------

// Default constructor
array::Parcel::Parcel(void) {}

// Constructor from CPU array
array::Parcel::Parcel(const array::Array & cpu_array, std::uintptr_t stream) : array::NdData(cpu_array) {
    // get device id
    this->device_ = cuda::Device();
    // allocate data
    hipError_t err_ = hipMalloc(&(this->data_), sizeof(float) * this->size());
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // cast stream
    hipStream_t copy_stream = reinterpret_cast<hipStream_t>(stream);
    // reset strides vector
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(float));
    // create copy function
    auto copy_func = std::bind(hipMemcpyAsync, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3,
                               hipMemcpyHostToDevice, copy_stream);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&cpu_array), copy_func);
}

// Constructor from a slice
array::Parcel::Parcel(const array::Parcel & whole, std::initializer_list<array::Slice> slices) :
        array::NdData(whole, slices) {
    this->force_free = false;
}

// Copy constructor
array::Parcel::Parcel(const array::Parcel & src) : array::NdData(src) {
    // get device id
    this->device_ = cuda::Device();
    // reform strides vector
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(float));
    // allocate data
    hipError_t err_ = hipMalloc(&(this->data_), sizeof(float) * this->size());
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // create copy function
    auto copy_func = std::bind(hipMemcpyPeer, std::placeholders::_1, this->device_.id(),
                               std::placeholders::_2, src.device_.id(), std::placeholders::_3);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
}

// Copy assignement
array::Parcel & array::Parcel::operator=(const array::Parcel & src) {
    // free old data
    this->free_current_data();
    // copy metadata and reform strides vector
    this->array::NdData::operator=(src);
    this->strides_ = array::contiguous_strides(this->shape_, sizeof(float));
    // allocate data
    hipError_t err_ = hipMalloc(&(this->data_), sizeof(float) * this->size());
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "Memory allocation failed with message \"%s\".\n", hipGetErrorString(err_));
    }
    // create copy function
    auto copy_func = std::bind(hipMemcpyPeer, std::placeholders::_1, this->device_.id(),
                               std::placeholders::_2, src.device_.id(), std::placeholders::_3);
    // copy data to GPU
    array::array_copy(dynamic_cast<array::NdData *>(this), dynamic_cast<const array::NdData *>(&src), copy_func);
    return *this;
}

// Move constructor
array::Parcel::Parcel(array::Parcel && src) : array::NdData(src) {
    // move device id
    this->device_ = src.device_;
    // take over pointer to source
    src.data_ = NULL;
}

// Move assignment
array::Parcel & array::Parcel::operator=(array::Parcel && src) {
    // free old data
    this->free_current_data();
    // move device id
    this->device_ = src.device_;
    // copy metadata
    this->array::NdData::operator=(src);
    // take over pointer to source
    src.data_ = NULL;
    return *this;
}

// Copy data to a pre-allocated memory
void array::Parcel::copy_to_gpu(array::Parcel * gpu_ptr, void * shape_strides_ptr) {
    // initialize buffer to store data of the copy before cloning it to GPU
    array::Parcel copy_on_gpu;
    // shallow copy of the current object
    copy_on_gpu.data_ = this->data_;
    copy_on_gpu.ndim_ = this->ndim_;
    copy_on_gpu.device_ = this->device_;
    // copy temporary object to GPU
    hipMemcpy(gpu_ptr, &copy_on_gpu, sizeof(array::Parcel), hipMemcpyHostToDevice);
    // copy shape and strides data
    this->shape_.copy_to_gpu(&(gpu_ptr->shape_), reinterpret_cast<std::uint64_t *>(shape_strides_ptr));
    this->strides_.copy_to_gpu(&(gpu_ptr->strides_), reinterpret_cast<std::uint64_t *>(shape_strides_ptr)+this->ndim_);
    // nullify data pointer to avoid free data
    copy_on_gpu.data_ = NULL;
    copy_on_gpu.shape_.data() = NULL;
    copy_on_gpu.strides_.data() = NULL;
}

// Free old data
void array::Parcel::free_current_data(void) {
    // lock mutex
    array::Parcel::m_.lock();
    // save current device and set device to the corresponding GPU
    cuda::Device current_device = cuda::Device::get_current_gpu();
    this->device_.set_as_current();
    // free data
    if ((this->data_ != NULL) && this->force_free) {
        hipFree(this->data_);
        this->data_ = NULL;
    }
    // finalize: set back the original GPU and unlock the mutex
    current_device.set_as_current();
    array::Parcel::m_.unlock();
}

// Destructor
array::Parcel::~Parcel(void) {
    this->free_current_data();
}

}  // namespace merlin
