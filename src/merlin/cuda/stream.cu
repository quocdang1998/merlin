#include "hip/hip_runtime.h"
// Copyright 2022 quocdang1998
#include "merlin/cuda/stream.hpp"

#include "merlin/logger.hpp"  // cuda_runtime_error, FAILURE, WARNING

namespace merlin {

// Constructor from setting flag and priority
cuda::Stream::Stream(cuda::Context & context, cuda::Stream::Setting setting, int priority) {
    // get min and max priority
    int min_priority, max_priority;
    ::hipDeviceGetStreamPriorityRange(&min_priority, &max_priority);
    if ((priority > min_priority) || (priority < max_priority)) {
        WARNING("Priority out of range (expected priority in range [%d, %d], got %d), the priority will be clamped.",
                max_priority, min_priority, priority);
    }
    // create a stream within the context
    hipStream_t stream;
    bool is_current = true;
    if (!context.is_current()) {
        is_current = false;
        context.push_current();
    }
    hipError_t err_ = ::hipStreamCreateWithPriority(&stream, static_cast<unsigned int>(setting), priority);
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipStreamCreate failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    this->stream_ = reinterpret_cast<std::uintptr_t>(stream);
    this->context_ = &context;
    if (!is_current) {
        context.pop_current();
    }
}

// Get flag
cuda::Stream::Setting cuda::Stream::setting(void) {
    unsigned int flag;
    hipError_t err_ = ::hipStreamGetFlags(reinterpret_cast<hipStream_t>(this->stream_), &flag);
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipStreamGetFlags failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    return static_cast<cuda::Stream::Setting>(flag);
}

// Get priority
int cuda::Stream::priority(void) {
    int priority;
    hipError_t err_ = ::hipStreamGetPriority(reinterpret_cast<hipStream_t>(this->stream_), &priority);
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipStreamGetPriority failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    return priority;
}

// Query for completion status
bool cuda::Stream::is_complete(void) {
    hipError_t err_ = ::hipStreamQuery(reinterpret_cast<hipStream_t>(this->stream_));
    if (err_ == hipSuccess) {
        return true;
    } else if (err_ != hipErrorNotReady) {
        FAILURE(cuda_runtime_error, "hipStreamQuery failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    return false;
}

// Add callback to stream
void cuda::Stream::launch_cpu_function(CudaStreamCallback func, void * arg) {
    hipLaunchHostFunc(reinterpret_cast<hipStream_t>(this->stream_), reinterpret_cast<hipHostFn_t>(func), arg);
}

// Synchronize the stream
void cuda::Stream::synchronize(void) {
    hipError_t err_ = ::hipStreamSynchronize(reinterpret_cast<hipStream_t>(this->stream_));
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipStreamSynchronize failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
}

// Destructor
cuda::Stream::~Stream(void) {
    hipError_t err_ = ::hipStreamDestroy(reinterpret_cast<hipStream_t>(this->stream_));
    if (err_ != hipSuccess) {
        FAILURE(cuda_runtime_error, "hipStreamDestroy failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
}

}  // namespace merlin
