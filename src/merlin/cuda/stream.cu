#include "hip/hip_runtime.h"
// Copyright 2022 quocdang1998
#include "merlin/cuda/stream.hpp"

#include <algorithm>  // std::clamp

#include "merlin/cuda/event.hpp"  // merlin::cuda::Event
#include "merlin/cuda/graph.hpp"  // merlin::cuda::Graph
#include "merlin/logger.hpp"      // merlin::Fatal, merlin::Warning, merlin::cuda_runtime_error

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Utils
// ---------------------------------------------------------------------------------------------------------------------

// Wrapper of the function adding CUDA callback to stream
void cuda::cuda_stream_add_callback(std::uintptr_t stream, cuda::StreamCallback func, void * arg) {
    ::hipError_t err_ = ::hipStreamAddCallback(reinterpret_cast<::hipStream_t>(stream), func, arg, 0);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add callback to stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// ---------------------------------------------------------------------------------------------------------------------
// Stream
// ---------------------------------------------------------------------------------------------------------------------

// Default constructor (the null stream)
cuda::Stream::Stream(void) { this->device_ = cuda::Device::get_current_gpu(); }

// Constructor from setting flag and priority
cuda::Stream::Stream(cuda::StreamSetting setting, int priority) {
    // get min and max priority
    int min_priority, max_priority;
    ::hipDeviceGetStreamPriorityRange(&min_priority, &max_priority);
    if ((priority > min_priority) || (priority < max_priority)) {
        Warning("Priority out of range (expected priority in range [%d, %d], got %d), the priority will be clamped.",
                max_priority, min_priority, priority);
    }
    priority = std::clamp(priority, min_priority, max_priority);
    // create a stream within the context
    ::hipStream_t stream;
    ::hipError_t err_ = ::hipStreamCreateWithPriority(&stream, static_cast<unsigned int>(setting), priority);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Create stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    this->stream_ = reinterpret_cast<std::uintptr_t>(stream);
    this->device_ = cuda::Device::get_current_gpu();
}

// Get flag
cuda::StreamSetting cuda::Stream::get_setting(void) const {
    unsigned int flag;
    ::hipError_t err_ = ::hipStreamGetFlags(reinterpret_cast<::hipStream_t>(this->stream_), &flag);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get flag of stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return static_cast<cuda::StreamSetting>(flag);
}

// Get priority
int cuda::Stream::get_priority(void) const {
    int priority;
    ::hipError_t err_ = ::hipStreamGetPriority(reinterpret_cast<::hipStream_t>(this->stream_), &priority);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get priority of stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return priority;
}

// Query for completion status
bool cuda::Stream::is_complete(void) const {
    ::hipError_t err_ = ::hipStreamQuery(reinterpret_cast<::hipStream_t>(this->stream_));
    if (err_ == 0) {
        return true;
    } else if (err_ != 600) {
        Fatal<cuda_runtime_error>("Query stream status failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return false;
}

// Check if the stream is being captured
bool cuda::Stream::is_capturing(void) const {
    ::hipStreamCaptureStatus capture_status;
    ::hipError_t err_ = ::hipStreamIsCapturing(reinterpret_cast<::hipStream_t>(this->stream_), &capture_status);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Query stream capture status failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    if (capture_status == 2) {
        Warning("Stream is capturing, but end capture is not beeing called.\n");
    }
    if (capture_status > 0) {
        return true;
    }
    return false;
}

// Check valid GPU and context
void cuda::Stream::check_cuda_context(void) const {
    if ((this->device_ != cuda::Device::get_current_gpu()) && (this->stream_ != 0)) {
        Fatal<cuda_runtime_error>("Current GPU is not the one associated the stream.\n");
    }
}

// Record event on a stream
void cuda::Stream::record_event(const cuda::Event & event) const {
    this->check_cuda_context();
    event.check_cuda_context();
    ::hipError_t err_ = ::hipEventRecord(reinterpret_cast<::hipEvent_t>(event.get_event_ptr()),
                                           reinterpret_cast<::hipStream_t>(this->stream_));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Record event failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// Wait on an event
void cuda::Stream::wait_event(const cuda::Event & event, cuda::EventWaitFlag flag) const {
    ::hipError_t err_ = ::hipStreamWaitEvent(reinterpret_cast<::hipStream_t>(this->stream_),
                                               reinterpret_cast<::hipEvent_t>(event.get_event_ptr()),
                                               static_cast<unsigned int>(flag));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Record event failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// Synchronize the stream
void cuda::Stream::synchronize(void) const {
    ::hipError_t err_ = ::hipStreamSynchronize(reinterpret_cast<::hipStream_t>(this->stream_));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Stream synchronization failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// Destructor
cuda::Stream::~Stream(void) {
    if (this->stream_ != 0) {
        hipError_t err_ = ::hipStreamDestroy(reinterpret_cast<hipStream_t>(this->stream_));
        if (err_ != 0) {
            Fatal<cuda_runtime_error>("hipStreamDestroy failed with message \"%s\".\n", ::hipGetErrorString(err_));
        }
    }
}

// ---------------------------------------------------------------------------------------------------------------------
// Stream capturing
// ---------------------------------------------------------------------------------------------------------------------

// Capturing stream for CUDA graph
void cuda::begin_capture_stream(const cuda::Stream & stream, StreamCaptureMode mode) {
    if (stream.is_capturing()) {
        Fatal<cuda_runtime_error>("Cannot re-capture a capturing stream.\n");
    }
    ::hipError_t err_ = ::hipStreamBeginCapture(reinterpret_cast<::hipStream_t>(stream.get_stream_ptr()),
                                                  static_cast<::hipStreamCaptureMode>(mode));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Capture stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// End capturing a stream and returning a graph
cuda::Graph cuda::end_capture_stream(const cuda::Stream & stream) {
    ::hipGraph_t graph_ptr;
    ::hipError_t err_ = ::hipStreamEndCapture(reinterpret_cast<::hipStream_t>(stream.get_stream_ptr()), &graph_ptr);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Retunr graph from captured stream failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return cuda::Graph(reinterpret_cast<std::uintptr_t>(graph_ptr));
}

}  // namespace merlin
