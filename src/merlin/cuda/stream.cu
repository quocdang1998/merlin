#include "hip/hip_runtime.h"
// Copyright 2022 quocdang1998
#include "merlin/cuda/stream.hpp"

#include <hip/hip_runtime.h>  // ::cuStreamGetCtx, ::hipCtx_t, ::hipStream_t

#include "merlin/cuda/context.hpp"  // merlin::cuda::Context
#include "merlin/cuda/event.hpp"  // merlin::cuda::Event
#include "merlin/cuda/graph.hpp"  // merlin::cuda::Graph
#include "merlin/logger.hpp"  // cuda_runtime_error, FAILURE, WARNING

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Stream
// --------------------------------------------------------------------------------------------------------------------

// Default constructor (the null stream)
cuda::Stream::Stream(void) {
    this->device_ = cuda::Device::get_current_gpu();
}

// Constructor from setting flag and priority
cuda::Stream::Stream(cuda::StreamSetting setting, int priority) {
    // get min and max priority
    int min_priority, max_priority;
    ::hipDeviceGetStreamPriorityRange(&min_priority, &max_priority);
    if ((priority > min_priority) || (priority < max_priority)) {
        WARNING("Priority out of range (expected priority in range [%d, %d], got %d), the priority will be clamped.",
                max_priority, min_priority, priority);
    }
    // create a stream within the context
    ::hipStream_t stream;
    ::hipError_t err_ = ::hipStreamCreateWithPriority(&stream, static_cast<unsigned int>(setting), priority);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    this->stream_ = reinterpret_cast<std::uintptr_t>(stream);
    this->device_ = cuda::Context::get_gpu_of_current_context();
}

// Get flag
cuda::StreamSetting cuda::Stream::setting(void) const {
    unsigned int flag;
    ::hipError_t err_ = ::hipStreamGetFlags(reinterpret_cast<::hipStream_t>(this->stream_), &flag);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get flag of stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return static_cast<cuda::StreamSetting>(flag);
}

// Get priority
int cuda::Stream::priority(void) const {
    int priority;
    ::hipError_t err_ = ::hipStreamGetPriority(reinterpret_cast<::hipStream_t>(this->stream_), &priority);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get priority of stream failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return priority;
}

// Get context associated to stream
cuda::Context cuda::Stream::get_context(void) const {
    ::hipCtx_t context;
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(this->stream_);
    ::hipError_t err_ = static_cast<::hipError_t>(::cuStreamGetCtx(stream, &context));
    return cuda::Context(reinterpret_cast<std::uintptr_t>(context));
}

// Query for completion status
bool cuda::Stream::is_complete(void) const {
    ::hipError_t err_ = ::hipStreamQuery(reinterpret_cast<::hipStream_t>(this->stream_));
    if (err_ == 0) {
        return true;
    } else if (err_ != 600) {
        FAILURE(cuda_runtime_error, "Query stream status failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return false;
}

// Check if the stream is being captured
bool cuda::Stream::is_capturing(void) const {
    ::hipStreamCaptureStatus capture_status;
    ::hipError_t err_ = ::hipStreamIsCapturing(reinterpret_cast<::hipStream_t>(this->stream_), &capture_status);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Query stream capture status failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    if (capture_status == 2) {
        WARNING("Stream is capturing, but end capture is not beeing called.\n");
    }
    if (capture_status > 0) {
        return true;
    }
    return false;
}

// Check valid GPU and context
void cuda::Stream::check_cuda_context(void) const {
    if (this->get_context() != cuda::Context::get_current()) {
        FAILURE(cuda_runtime_error, "Current context is not the one associated the stream.\n");
    }
    if (this->device_ != cuda::Context::get_gpu_of_current_context()) {
        FAILURE(cuda_runtime_error, "Current GPU is not the one associated the stream.\n");
    }
}

// Add callback to stream
void cuda::Stream::add_callback(cuda::Stream::CudaStreamCallback func, void * arg) const {
    ::hipError_t err_ = ::hipStreamAddCallback(reinterpret_cast<::hipStream_t>(this->stream_), func, arg, 0);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Add callback to stream failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
}

// Record event on a stream
void cuda::Stream::record_event(const cuda::Event & event) const {
    this->check_cuda_context();
    event.check_cuda_context();
    ::hipError_t err_ = ::hipEventRecord(reinterpret_cast<::hipEvent_t>(event.get_event_ptr()),
                                           reinterpret_cast<::hipStream_t>(this->stream_));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Record event failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// Wait on an event
void cuda::Stream::wait_event(const cuda::Event & event, cuda::EventWaitFlag flag) const {
    ::hipError_t err_ = ::hipStreamWaitEvent(reinterpret_cast<::hipStream_t>(this->stream_),
                                               reinterpret_cast<::hipEvent_t>(event.get_event_ptr()),
                                               static_cast<unsigned int>(flag));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Record event failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// Synchronize the stream
void cuda::Stream::synchronize(void) const {
    ::hipError_t err_ = ::hipStreamSynchronize(reinterpret_cast<::hipStream_t>(this->stream_));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Stream synchronization failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
}

// Destructor
cuda::Stream::~Stream(void) {
    if (this->stream_ != 0) {
        hipError_t err_ = ::hipStreamDestroy(reinterpret_cast<hipStream_t>(this->stream_));
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "hipStreamDestroy failed with message \"%s\".\n", ::hipGetErrorString(err_));
        }
    }
}

// Capturing stream for CUDA graph
void cuda::begin_capture_stream(const cuda::Stream & stream, StreamCaptureMode mode) {
    if (stream.is_capturing()) {
        FAILURE(cuda_runtime_error, "Cannot re-capture a capturing stream.\n");
    }
    ::hipError_t err_ = ::hipStreamBeginCapture(reinterpret_cast<::hipStream_t>(stream.get_stream_ptr()),
                                                  static_cast<::hipStreamCaptureMode>(mode));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Capture stream failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
}

// End capturing a stream and returning a graph
cuda::Graph cuda::end_capture_stream(const cuda::Stream & stream) {
    ::hipGraph_t graph_ptr;
    ::hipError_t err_ = ::hipStreamEndCapture(reinterpret_cast<::hipStream_t>(stream.get_stream_ptr()), &graph_ptr);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Retunr graph from captured stream failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return cuda::Graph(reinterpret_cast<std::uintptr_t>(graph_ptr));
}

}  // namespace merlin
