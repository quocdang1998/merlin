// Copyright 2022 quocdang1998
#include "merlin/cuda/graph.hpp"

#include <cstddef>  // std::size_t
#include <cstring>  // std::memset

#include "merlin/cuda/device.hpp"  // merlin::cuda::Device
#include "merlin/cuda/event.hpp"   // merlin::cuda::Event
#include "merlin/cuda/stream.hpp"  // merlin::cuda::Stream
#include "merlin/logger.hpp"       // merlin::Fatal, merlin::Warning, merlin::cuda_runtime_error

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Utility
// ---------------------------------------------------------------------------------------------------------------------

// Wrapper of the function adding CUDA callback to graph
cuda::GraphNode cuda::add_callback_to_graph(std::uintptr_t graph_ptr, cuda::GraphCallback functor,
                                            const cuda::GraphNodeList & deps, void * arg) {
    ::hipGraphNode_t graph_node;
    ::hipHostNodeParams function_params;
    function_params.fn = functor;
    function_params.userData = arg;
    ::hipError_t err_ = ::hipGraphAddHostNode(&graph_node, reinterpret_cast<::hipGraph_t>(graph_ptr),
                                                reinterpret_cast<const ::hipGraphNode_t *>(deps.data()), deps.size(),
                                                &function_params);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add host function node to graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node));
}

// ---------------------------------------------------------------------------------------------------------------------
// GraphNode
// ---------------------------------------------------------------------------------------------------------------------

// Get node type
cuda::NodeType cuda::GraphNode::get_node_type(void) const {
    ::hipGraphNodeType type;
    ::hipError_t err_ = ::hipGraphNodeGetType(reinterpret_cast<::hipGraphNode_t>(this->node_id), &type);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("CUDA get node type failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return static_cast<cuda::NodeType>(type);
}

// ---------------------------------------------------------------------------------------------------------------------
// Graph
// ---------------------------------------------------------------------------------------------------------------------

// Destroy current CUDA graph instance
void cuda::Graph::destroy_graph(void) {
    if (this->graph_ != 0) {
        ::hipError_t err_ = ::hipGraphDestroy(reinterpret_cast<::hipGraph_t>(this->graph_));
        if (err_ != 0) {
            Fatal<cuda_runtime_error>("CUDA destroy graph failed with message \"%s\".\n", ::hipGetErrorString(err_));
        }
        this->graph_ = 0;
    }
}

// Constructor
cuda::Graph::Graph(int flag) {
    ::hipError_t err_;
    switch (flag) {
        case -1 : {  // default constructor
            break;
        }
        case 0 : {  // construct an empty graph
            ::hipGraph_t graph_;
            err_ = ::hipGraphCreate(&graph_, 0);
            if (err_ != 0) {
                Fatal<cuda_runtime_error>("CUDA create graph failed with message \"%s\".\n",
                                          ::hipGetErrorString(err_));
            }
            this->graph_ = reinterpret_cast<std::uintptr_t>(graph_);
            break;
        }
        default : {  // error unknown argument
            Fatal<std::invalid_argument>("Expected 0 (new empty graph) or -1 (NULL graph), got %d.\n", flag);
            break;
        }
    }
}

// Copy constructor
cuda::Graph::Graph(const cuda::Graph & src) {
    ::hipGraph_t graph_, graph_src = reinterpret_cast<::hipGraph_t>(src.graph_);
    ::hipError_t err_ = ::hipGraphClone(&graph_, graph_src);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("CUDA clone graph failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    this->graph_ = reinterpret_cast<std::uintptr_t>(graph_);
}

// Copy assignment
cuda::Graph & cuda::Graph::operator=(const cuda::Graph & src) {
    // Destroy current isntance
    this->destroy_graph();
    // Clone graph
    ::hipGraph_t graph_, graph_src = reinterpret_cast<::hipGraph_t>(src.graph_);
    ::hipError_t err_ = ::hipGraphClone(&graph_, graph_src);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("CUDA clone graph failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    this->graph_ = reinterpret_cast<std::uintptr_t>(graph_);
    return *this;
}

// Move constructor
cuda::Graph::Graph(cuda::Graph && src) {
    this->graph_ = src.graph_;
    src.graph_ = 0;
}

// Move assignment
cuda::Graph & cuda::Graph::operator=(cuda::Graph && src) {
    // Destroy current isntance
    this->destroy_graph();
    // Move graph pointer
    this->graph_ = src.graph_;
    src.graph_ = 0;
    return *this;
}

// Get number of nodes in a graph
std::uint64_t cuda::Graph::get_num_nodes(void) const {
    std::size_t num_nodes;
    ::hipError_t err_ = ::hipGraphGetNodes(reinterpret_cast<::hipGraph_t>(this->graph_), nullptr, &num_nodes);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get number of nodes of CUDA graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return std::uint64_t(num_nodes);
}

// Get node list
cuda::GraphNodeList cuda::Graph::get_node_list(void) const {
    std::size_t num_nodes = this->get_num_nodes();
    cuda::GraphNodeList node_list(num_nodes);
    ::hipError_t err_ = ::hipGraphGetNodes(reinterpret_cast<::hipGraph_t>(this->graph_),
                                             reinterpret_cast<::hipGraphNode_t *>(node_list.data()), &num_nodes);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get node list of CUDA graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return node_list;
}

// Get number of edges
std::uint64_t cuda::Graph::get_num_edges(void) const {
    std::size_t num_edges;
    ::hipError_t err_ = ::hipGraphGetEdges(reinterpret_cast<::hipGraph_t>(this->graph_), nullptr, nullptr,
                                             &num_edges);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get number of edges of CUDA graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return std::uint64_t(num_edges);
}

// Get edge list
cuda::GraphEdgeList cuda::Graph::get_edge_list(void) const {
    // allocate memory
    std::size_t num_edges = this->get_num_edges();
    vector::DynamicVector<::hipGraphNode_t> nodes_from(num_edges), nodes_to(num_edges);
    // get edge list
    ::hipError_t err_ = ::hipGraphGetEdges(reinterpret_cast<::hipGraph_t>(this->graph_), nodes_from.data(),
                                             nodes_to.data(), &num_edges);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get edge list of CUDA graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    // transform result
    cuda::GraphEdgeList edge_list(num_edges);
    for (std::uint64_t i = 0; i < num_edges; i++) {
        edge_list[i][0].node_id = reinterpret_cast<std::uintptr_t>(nodes_from[i]);
        edge_list[i][1].node_id = reinterpret_cast<std::uintptr_t>(nodes_to[i]);
    }
    return edge_list;
}

// Add memory allocation node
std::pair<cuda::GraphNode, void *> cuda::Graph::add_malloc_node(std::uint64_t size, const cuda::GraphNodeList & deps) {
    ::hipGraphNode_t graph_node;
    ::hipMemAllocNodeParams node_params;
    std::memset(&node_params, 0, sizeof(::hipMemAllocNodeParams));
    node_params.bytesize = size;
    node_params.poolProps.allocType = ::hipMemAllocationTypePinned;
    node_params.poolProps.location.id = cuda::Device::get_current_gpu().id();
    node_params.poolProps.location.type = ::hipMemLocationTypeDevice;
    ::hipError_t err_ = ::hipGraphAddMemAllocNode(&graph_node, reinterpret_cast<::hipGraph_t>(this->graph_),
                                                    reinterpret_cast<const ::hipGraphNode_t *>(deps.data()),
                                                    deps.size(), &node_params);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add memory allocation node to graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return std::pair<cuda::GraphNode, void *>(cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node)),
                                              node_params.dptr);
}

// Add memcpy node
cuda::GraphNode cuda::Graph::add_memcpy_node(void * dest, const void * src, std::uint64_t size,
                                             cuda::MemcpyKind copy_flag, const cuda::GraphNodeList & deps) {
    ::hipGraphNode_t graph_node;
    ::hipError_t err_ = ::hipGraphAddMemcpyNode1D(&graph_node, reinterpret_cast<::hipGraph_t>(this->graph_),
                                                    reinterpret_cast<const ::hipGraphNode_t *>(deps.data()),
                                                    deps.size(), dest, src, size,
                                                    static_cast<::hipMemcpyKind>(copy_flag));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add memcpy node to graph failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    return cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node));
}

// Add CUDA deallocation node
cuda::GraphNode cuda::Graph::add_memfree_node(void * ptr, const cuda::GraphNodeList & deps) {
    ::hipGraphNode_t graph_node;
    ::hipError_t err_ = ::hipGraphAddMemFreeNode(&graph_node, reinterpret_cast<::hipGraph_t>(this->graph_),
                                                   reinterpret_cast<const ::hipGraphNode_t *>(deps.data()),
                                                   deps.size(), ptr);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add memfree node to graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node));
}

// Add CUDA event record node
cuda::GraphNode cuda::Graph::add_event_record_node(const cuda::Event & event, const cuda::GraphNodeList & deps) {
    ::hipGraphNode_t graph_node;
    ::hipError_t err_ = ::hipGraphAddEventRecordNode(&graph_node, reinterpret_cast<::hipGraph_t>(this->graph_),
                                                       reinterpret_cast<const ::hipGraphNode_t *>(deps.data()),
                                                       deps.size(),
                                                       reinterpret_cast<::hipEvent_t>(event.get_event_ptr()));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add event record node to graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node));
}

// Add CUDA event wait node
cuda::GraphNode cuda::Graph::add_event_wait_node(const cuda::Event & event, const cuda::GraphNodeList & deps) {
    ::hipGraphNode_t graph_node;
    ::hipError_t err_ = ::hipGraphAddEventWaitNode(&graph_node, reinterpret_cast<::hipGraph_t>(this->graph_),
                                                     reinterpret_cast<const ::hipGraphNode_t *>(deps.data()),
                                                     deps.size(),
                                                     reinterpret_cast<::hipEvent_t>(event.get_event_ptr()));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add event wait node to graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node));
}

// Add CUDA child graph node
cuda::GraphNode cuda::Graph::add_child_graph_node(const cuda::Graph & child_graph, const cuda::GraphNodeList & deps) {
    ::hipGraphNode_t graph_node;
    ::hipError_t err_ = ::hipGraphAddChildGraphNode(&graph_node, reinterpret_cast<::hipGraph_t>(this->graph_),
                                                      reinterpret_cast<const ::hipGraphNode_t *>(deps.data()),
                                                      deps.size(), reinterpret_cast<::hipGraph_t>(child_graph.graph_));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Add child graph node to graph failed with message \"%s\".\n",
                                  ::hipGetErrorString(err_));
    }
    return cuda::GraphNode(reinterpret_cast<std::uintptr_t>(graph_node));
}

// Export graph into DOT file
void cuda::Graph::export_to_dot(const std::string & filename) {
    ::hipError_t err_ = ::hipGraphDebugDotPrint(reinterpret_cast<::hipGraph_t>(this->graph_), filename.c_str(), 0);
}

// Execute a graph (add detecting errored node)
void cuda::Graph::execute(const cuda::Stream & stream) {
    ::hipGraphExec_t exec_graph;
    char log_buffer[256];
    std::memset(log_buffer, 0, sizeof(log_buffer));
    ::hipError_t err_ = ::hipGraphInstantiate(&exec_graph, reinterpret_cast<::hipGraph_t>(this->graph_), nullptr,
                                                log_buffer, sizeof(log_buffer));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Create executable graph failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    if (log_buffer[0]) {  // not a null started string
        Warning("Instantiate graph executable failed with error \"%s\"\n", log_buffer);
    }
    cuda::CtxGuard guard(stream.get_gpu());
    err_ = ::hipGraphLaunch(exec_graph, reinterpret_cast<::hipStream_t>(stream.get_stream_ptr()));
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Launch graph failed with error: \"%s\"\n", ::hipGetErrorString(err_));
    }
}

// Destructor
cuda::Graph::~Graph(void) { this->destroy_graph(); }

}  // namespace merlin
