// Copyright 2022 quocdang1998
#include "merlin/cuda/event.hpp"

#include "merlin/logger.hpp"  // cuda_runtime_error, FAILURE, WARNING

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// Event
// --------------------------------------------------------------------------------------------------------------------

// Contruct an event with a given flag
cuda::Event::Event(cuda::Event::Category category) : category_(category), device_(cuda::Device::get_current_gpu()),
context_(cuda::Context::get_current()) {
    ::hipEvent_t event;
    ::hipError_t err_ = ::hipEventCreateWithFlags(&event, static_cast<unsigned int>(category));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create event failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    this->event_ = reinterpret_cast<std::uintptr_t>(event);
}

// Query the status of works
bool cuda::Event::is_complete(void) const {
    ::hipError_t err_ = ::hipEventQuery(reinterpret_cast<::hipEvent_t>(this->event_));
    if ((err_ != 0) && (err_ != 600)) {
        FAILURE(cuda_runtime_error, "Query event failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
    if (err_ == 0) {
        return true;
    }
    return false;
}

// Check valid GPU and context
void cuda::Event::check_cuda_context(void) const {
    if (this->context_ != cuda::Context::get_current()) {
        FAILURE(cuda_runtime_error, "Current context is not the one associated the event.\n");
    }
    if (this->device_ != cuda::Context::get_gpu_of_current_context()) {
        FAILURE(cuda_runtime_error, "Current GPU is not the one associated the event.\n");
    }
}

// Synchronize the event
void cuda::Event::synchronize(void) const {
    ::hipError_t err_ = ::hipEventSynchronize(reinterpret_cast<::hipEvent_t>(this->event_));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Event synchronization failed with message \"%s\".\n", ::hipGetErrorName(err_));
    }
}

// Get elapsed time between 2 events
float cuda::operator-(const cuda::Event & ev_1, const cuda::Event & ev_2) {
    float result;
    ::hipError_t err_ = ::hipEventElapsedTime(&result, reinterpret_cast<::hipEvent_t>(ev_1.event_),
                                                reinterpret_cast<::hipEvent_t>(ev_2.event_));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Calculate elapsed time between 2 event failed with message \"%s\".\n",
                ::hipGetErrorName(err_));
    }
    return result;
}

// Destructor
cuda::Event::~Event(void) {
    if (this->event_ != 0) {
        ::hipError_t err_ = ::hipEventDestroy(reinterpret_cast<::hipEvent_t>(this->event_));
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "Destroy event failed with message \"%s\".\n", ::hipGetErrorName(err_));
        }
    }
}

}  // namespace merlin
