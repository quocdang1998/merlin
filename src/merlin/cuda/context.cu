// Copyright 2022 quocdang1998
#include "merlin/cuda/context.hpp"

#include <utility>  // std::pair

#include <hip/hip_runtime.h>  // ::hipCtxCreate, ::hipCtxDestroy, ::hipCtxGetCurrent, ::hipCtxGetDevice, ::hipCtxGetFlags,
                   // ::hipCtxPushCurrent, ::hipCtxPopCurrent, ::hipCtxSynchronize, ::hipCtx_t,
                   // ::hipDevicePrimaryCtxGetState, ::hipDevicePrimaryCtxRelease, ::hipDevicePrimaryCtxRetain

#include "merlin/cuda/device.hpp"  // merlin::cuda::Device
#include "merlin/logger.hpp"       // cuda_runtime_error, FAILURE

#define safety_lock() bool lock_success = Environment::mutex.try_lock()
#define safety_unlock()                                                                                                \
    if (lock_success) Environment::mutex.unlock()

namespace merlin {

// ----------------------------------------------------------------------------------------------------------------------
// CUDA Context
// ----------------------------------------------------------------------------------------------------------------------

// Get pointer to current context
static inline std::uintptr_t get_current_context_ptr(void) {
    // check for current context as regular context
    safety_lock();
    ::hipCtx_t current_ctx;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxGetCurrent(&current_ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    // return if current context is a regular one
    if (current_ctx != nullptr) {
        return reinterpret_cast<std::uintptr_t>(current_ctx);
    }
    // if null pointer returned, query for primary context
    int num_gpu = cuda::Device::get_num_gpu();
    for (int i_gpu = 0; i_gpu < num_gpu; i_gpu++) {
        unsigned int flag;
        int active = 0;
        err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxGetState(i_gpu, &flag, &active));
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "Query primary context for GPU ID %d failed with message \"%s\".\n", i_gpu,
                    ::hipGetErrorString(err_));
        }
        if (active) {
            err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxRetain(&current_ctx, i_gpu));
            if (err_ != 0) {
                FAILURE(cuda_runtime_error, "Get primary context for GPU ID %d failed with message \"%s\".\n", i_gpu,
                        ::hipGetErrorString(err_));
            }
            std::uintptr_t context_ptr = reinterpret_cast<std::uintptr_t>(current_ctx);
            Environment::primary_contexts[i_gpu] = context_ptr;
            return context_ptr;
        }
    }
    // a dummy context initialized (return nullptr)
    safety_unlock();
    return reinterpret_cast<std::uintptr_t>(current_ctx);
}

// ----------------------------------------------------------------------------------------------------------------------
// Context
// ----------------------------------------------------------------------------------------------------------------------

// Member constructor
cuda::Context::Context(const cuda::Device & gpu, cuda::ContextSchedule schedule) {
    // Create context
    safety_lock();
    ::hipCtx_t ctx;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxCreate(&ctx, static_cast<unsigned int>(schedule), gpu.id()));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    this->context_ = reinterpret_cast<std::uintptr_t>(ctx);
    // Increase reference count and initialize attached flag
    Environment::ContextAttribute attrib(1, cuda::Device::get_current_gpu().id());
    auto [it_current, success] = Environment::attribute.insert(std::pair(this->context_, attrib));
    if (!success) {
        FAILURE(cuda_runtime_error, "Create context failed because the context has already exist.\n");
    }
    safety_unlock();
}

// Constructor from context pointer (to be improved for the case of primary context)
cuda::Context::Context(std::uintptr_t context_ptr) : context_(context_ptr) {
    safety_lock();
    if (Environment::attribute.find(context_ptr) == Environment::attribute.end()) {
        Environment::attribute[context_ptr] = {0, cuda::Device::get_current_gpu().id()};
    }
    Environment::attribute[context_ptr].reference_count += 1;
    safety_unlock();
}

// Check if the context is the top of context stack
bool cuda::Context::is_current(void) const { return this->context_ == get_current_context_ptr(); }

// Push the context to the stack
void cuda::Context::push_current(void) const {
    safety_lock();
    ::hipCtx_t ctx = reinterpret_cast<::hipCtx_t>(this->context_);
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxPushCurrent(ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Push context to current stack failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    safety_unlock();
}

// Pop the context out of the stack
const cuda::Context & cuda::Context::pop_current(void) const {
    safety_lock();
    if (!(this->is_current())) {
        FAILURE(cuda_runtime_error, "The current context is not being attached to the current processes.\n");
    }
    ::hipCtx_t ctx = reinterpret_cast<::hipCtx_t>(this->context_);
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxPopCurrent(&ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Pop current context out of the stack failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    safety_unlock();
    return *this;
}

// Get current context
cuda::Context cuda::Context::get_current(void) {
    safety_lock();
    cuda::Context result;
    result.context_ = get_current_context_ptr();
    if (Environment::attribute.find(result.context_) == Environment::attribute.end()) {
        int current_gpu = cuda::Device::get_current_gpu().id();
        Environment::attribute[result.context_] = Environment::ContextAttribute(1, current_gpu);
    }
    Environment::attribute[result.context_].reference_count += 1;
    safety_unlock();
    return result;
}

// Get GPU attached to current context
cuda::Device cuda::Context::get_gpu_of_current_context(void) {
    safety_lock();
    int device;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxGetDevice(&device));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get GPU ID of current context failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    safety_unlock();
    return cuda::Device(device);
}

// Get flag of the current context.
cuda::ContextSchedule cuda::Context::get_flag_of_current_context(void) {
    safety_lock();
    unsigned int flag;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxGetFlags(&flag));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get flag of current context failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    safety_unlock();
    return static_cast<cuda::ContextSchedule>(flag);
}

// Synchronize current context
void cuda::Context::synchronize(void) {
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxSynchronize());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Synchronize current context failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
}

// Destructor
cuda::Context::~Context(void) {
    // free if the context is not a primary context and reference count goes to zero
    if (this->context_ != 0) {
        safety_lock();
        Environment::attribute[this->context_].reference_count -= 1;
        if (Environment::attribute[this->context_].reference_count == 0) {
            if (!this->is_primary()) {
                ::hipCtxDestroy(reinterpret_cast<::hipCtx_t>(this->context_));
            }
            Environment::attribute.erase(this->context_);
        }
        this->context_ = 0;
        safety_unlock();
    }
}

// Create a primary context attached to a GPU
cuda::Context cuda::create_primary_context(const cuda::Device & gpu, cuda::ContextSchedule flag) {
    // check validity of GPU
    if (gpu.id() < 0) {
        FAILURE(cuda_runtime_error, "Invalid GPU ID (id = %d).\n", gpu.id());
    }
    cuda::Context result;
    ::hipError_t err_;
    // find already initialized context
    for (auto & [ctx_ptr, attribute] : Environment::attribute) {
        if (attribute.gpu == gpu.id()) {
            result.context_ = ctx_ptr;
            attribute.reference_count += 1;
            err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxSetFlags(gpu.id(), static_cast<unsigned int>(flag)));
            if (err_ != 0) {
                FAILURE(cuda_runtime_error, "Set flag to primary context for GPU %d failed with message \"%s\".\n",
                        gpu.id(), ::hipGetErrorName(err_));
            }
            return result;
        }
    }
    // retain context if not initialized
    ::hipCtx_t ctx;
    err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxRetain(&ctx, gpu.id()));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create primary context for GPU %d failed with message \"%s\".\n", gpu.id(),
                ::hipGetErrorName(err_));
    }
    result.context_ = reinterpret_cast<std::uintptr_t>(ctx);
    Environment::attribute.insert({
        result.context_, {1, gpu.id()}
    });
    err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxSetFlags(gpu.id(), static_cast<unsigned int>(flag)));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Set flag to primary context for GPU %d failed with message \"%s\".\n", gpu.id(),
                ::hipGetErrorName(err_));
    }
    return result;
}

}  // namespace merlin
