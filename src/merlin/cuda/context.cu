// Copyright 2022 quocdang1998
#include "merlin/cuda/context.hpp"

#include <utility>  // std::pair

#include <hip/hip_runtime.h>  // ::hipCtxCreate, ::hipCtxDestroy, ::hipCtxGetCurrent, ::hipCtxGetDevice, ::hipCtxGetFlags,
                   // ::hipCtxPushCurrent, ::hipCtxPopCurrent, ::hipCtxSynchronize, ::hipCtx_t,
                   // ::hipDevicePrimaryCtxGetState, ::hipDevicePrimaryCtxRelease, ::hipDevicePrimaryCtxRetain

#include "merlin/cuda/device.hpp"  // merlin::cuda::Device

namespace merlin {

// --------------------------------------------------------------------------------------------------------------------
// CUDA Context
// --------------------------------------------------------------------------------------------------------------------

// Get pointer to current context
static inline std::uintptr_t get_current_context_ptr(void) {
    // check for current context as regular context
    ::hipCtx_t current_ctx;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxGetCurrent(&current_ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    // return if current context is a regular one
    if (current_ctx != nullptr) {
        return reinterpret_cast<std::uintptr_t>(current_ctx);
    }
    // if null pointer returned, query for primary context
    int num_gpu = cuda::Device::get_num_gpu();
    for (int i_gpu = 0; i_gpu < num_gpu; i_gpu++) {
        unsigned int flag;
        int active = 0;
        err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxGetState(i_gpu, &flag, &active));
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "Query primary context for GPU ID %d failed with message \"%s\".\n", i_gpu,
                    ::hipGetErrorString(err_));
        }
        if (active) {
            err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxRetain(&current_ctx, i_gpu));
            if (err_ != 0) {
                FAILURE(cuda_runtime_error, "Get primary context for GPU ID %d failed with message \"%s\".\n", i_gpu,
                        ::hipGetErrorString(err_));
            }
            return reinterpret_cast<std::uintptr_t>(current_ctx);
        }
    }
    // a dummy context initialized (return nullptr)
    WARNING("Current CUDA context is nullptr.\n");
    return reinterpret_cast<std::uintptr_t>(current_ctx);
}

// --------------------------------------------------------------------------------------------------------------------
// Context
// --------------------------------------------------------------------------------------------------------------------

// Member constructor
cuda::Context::Context(const cuda::Device & gpu, cuda::Context::Flags flag) {
    // Create context
    ::hipCtx_t ctx;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxCreate(&ctx, static_cast<unsigned int>(flag), gpu.id()));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    this->context_ = reinterpret_cast<std::uintptr_t>(ctx);
    // Increase reference count and initialize attached flag
    Environment::ContextAttribute attrib(1, cuda::Device::get_current_gpu().id());
    auto [it_current, success] = Environment::attribute.insert(std::pair(this->context_, attrib));
    if (!success) {
        FAILURE(cuda_runtime_error, "Create context failed because the context has already exist.\n");
    }
}

// Constructor from context pointer (to be improved for the case of primary context)
cuda::Context::Context(std::uintptr_t context_ptr) : context_(context_ptr) {
    if (Environment::attribute.find(context_ptr) == Environment::attribute.end()) {
        Environment::attribute[context_ptr] = {0, cuda::Device::get_current_gpu().id()};
    }
    Environment::attribute[context_ptr].reference_count += 1;
}

// Check if the context is the top of context stack
bool cuda::Context::is_current(void) {
    return this->context_ == get_current_context_ptr();
}

// Push the context to the stack
void cuda::Context::push_current(void) {
    if (this->is_current()) {
        FAILURE(cuda_runtime_error, "The current context is being attached to the CPU process.\n");
    }
    ::hipCtx_t ctx = reinterpret_cast<::hipCtx_t>(this->context_);
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxPushCurrent(ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Push context to current stack failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
}

// Pop the context out of the stack
cuda::Context & cuda::Context::pop_current(void) {
    if (!(this->is_current())) {
        FAILURE(cuda_runtime_error, "The current context is not being attached to any processes.\n");
    }
    ::hipCtx_t ctx = reinterpret_cast<::hipCtx_t>(this->context_);
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxPopCurrent(&ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Pop current context out of the stack failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return *this;
}

// Get current context
cuda::Context cuda::Context::get_current(void) {
    cuda::Context result;
    result.context_ = get_current_context_ptr();
    if (Environment::attribute.find(result.context_) == Environment::attribute.end()) {
        int current_gpu = cuda::Device::get_current_gpu().id();
        Environment::attribute[result.context_] = Environment::ContextAttribute(1, current_gpu);
    }
    Environment::attribute[result.context_].reference_count += 1;
    return result;
}

// Get GPU attached to current context
cuda::Device cuda::Context::get_gpu_of_current_context(void) {
    int device;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxGetDevice(&device));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get GPU ID of current context failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return cuda::Device(device);
}

// Get flag of the current context.
cuda::Context::Flags cuda::Context::get_flag_of_current_context(void) {
    unsigned int flag;
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxGetFlags(&flag));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get flag of current context failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
    return static_cast<cuda::Context::Flags>(flag);
}

// Synchronize current context
void cuda::Context::synchronize(void) {
    ::hipError_t err_ = static_cast<::hipError_t>(::hipCtxSynchronize());
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Synchronize current context failed with message \"%s\".\n",
                ::hipGetErrorString(err_));
    }
}

// Destructor
cuda::Context::~Context(void) {
    // free if the context is not a primary context and reference count goes to zero
    if (this->context_ != 0) {
        Environment::attribute[this->context_].reference_count -= 1;
        if (Environment::attribute[this->context_].reference_count == 0) {
            if (!this->is_primary()) {
                ::hipCtxDestroy(reinterpret_cast<::hipCtx_t>(this->context_));
            }
            Environment::attribute.erase(this->context_);
        }
    }
}

// Create a primary context attached to a GPU
cuda::Context cuda::create_primary_context(const cuda::Device & gpu, cuda::Context::Flags flag) {
    // check validity of GPU
    if (gpu.id() < 0) {
        FAILURE(cuda_runtime_error, "Invalid GPU ID (id = %d).\n", gpu.id());
    }
    cuda::Context result;
    ::hipError_t err_;
    // find already initialized context
    for (auto & [ctx_ptr, attribute] : Environment::attribute) {
        if (attribute.gpu == gpu.id()) {
            result.context_ = ctx_ptr;
            attribute.reference_count += 1;
            err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxSetFlags(gpu.id(), static_cast<unsigned int>(flag)));
            if (err_ != 0) {
                FAILURE(cuda_runtime_error, "Set flag to primary context for GPU %d failed with message \"%s\".\n",
                        gpu.id(), ::hipGetErrorName(err_));
            }
            return result;
        }
    }
    // retain context if not initialized
    ::hipCtx_t ctx;
    err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxRetain(&ctx, gpu.id()));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create primary context for GPU %d failed with message \"%s\".\n", gpu.id(),
                ::hipGetErrorName(err_));
    }
    result.context_ = reinterpret_cast<std::uintptr_t>(ctx);
    Environment::attribute.insert({result.context_, {1, gpu.id()}});
    err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxSetFlags(gpu.id(), static_cast<unsigned int>(flag)));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Set flag to primary context for GPU %d failed with message \"%s\".\n", gpu.id(),
                ::hipGetErrorName(err_));
    }
    return result;
}

}  // namespace merlin
