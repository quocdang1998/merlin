// Copyright 2022 quocdang1998
#include "merlin/cuda/context.hpp"

#include <sstream>  // std::ostringstream

#include "hip/hip_runtime.h"  // hipCtxCreate, hipCtxDestroy, hipCtx_t

namespace merlin::cuda {

// --------------------------------------------------------------------------------------------------------------------
// Context
// --------------------------------------------------------------------------------------------------------------------

// Member constructor
Context::Context(const Device & gpu, Context::Flags flag) {
    // Create context
    hipCtx_t ctx;
    hipError_t err_ = static_cast<hipError_t>(hipCtxCreate(&ctx, static_cast<unsigned int>(flag), gpu.id()));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Create context failed with message \"%s\".\n", hipGetErrorName(err_));
    }
    this->context_ = reinterpret_cast<std::uintptr_t>(ctx);
    // Increase reference count and initialize attached flag
    Context::m_.lock();
    auto [it_current, success] = Context::shared_attributes_.insert({this->context_, {1, true, gpu}});
    if (!success) {
        FAILURE(cuda_runtime_error, "Create context failed because the context has already exist.\n");
    }
    Context::m_.unlock();
}


// Push the context to the stack
void Context::push_current(void) {
    if (this->is_attached()) {
        FAILURE(cuda_runtime_error, "The current context is being attached to the CPU process\n");
    }
    hipCtx_t ctx = reinterpret_cast<hipCtx_t>(this->context_);
    hipError_t err_ = static_cast<hipError_t>(hipCtxPushCurrent(ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Push context to current stack failed with message \"%s\".\n",
                hipGetErrorName(err_));
    }
    Context::m_.lock();
    Context::shared_attributes_[this->context_].attached = true;
    Context::m_.unlock();
}

// Pop the context out of the stack
Context & Context::pop_current(void) {
    if (!(this->is_attached())) {
        FAILURE(cuda_runtime_error, "The current context is not being attached to any processes\n");
    }
    hipCtx_t ctx = reinterpret_cast<hipCtx_t>(this->context_);
    hipError_t err_ = static_cast<hipError_t>(hipCtxPopCurrent(&ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Pop current context out of the stack failed with message \"%s\".\n",
                hipGetErrorName(err_));
    }
    Context::m_.lock();
    Context::shared_attributes_[this->context_].attached = false;
    Context::m_.unlock();
    return *this;
}

// Get current context
Context Context::get_current(void) {
    Context result;
    hipCtx_t current_ctx;
    hipError_t err_ = static_cast<hipError_t>(hipCtxGetCurrent(&current_ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", hipGetErrorName(err_));
    }
    result.context_ = reinterpret_cast<std::uintptr_t>(current_ctx);
    Context::m_.lock();
    if (Context::shared_attributes_.find(result.context_) == Context::shared_attributes_.end()) {
        Context::shared_attributes_[result.context_] = {1, true, Device::get_current_gpu()};
    }
    Context::shared_attributes_[result.context_].reference_count += 1;
    Context::m_.unlock();
    return result;
}

// Check if the context is the top of context stack
bool Context::is_current(void) {
    Context current = Context::get_current();
    return (this->context_ == current.context_);
}

// Set current context at the top of the stack
void Context::set_current(void) {
    if (!(this->is_attached())) {
        FAILURE(cuda_runtime_error, "The current context is not being attached to any process\n");
    }
    hipCtx_t current_ctx = reinterpret_cast<hipCtx_t>(this->context_);
    hipError_t err_ = static_cast<hipError_t>(hipCtxSetCurrent(current_ctx));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Set current context failed with message \"%s\".\n", hipGetErrorName(err_));
    }
}

// String representation
std::string Context::repr(void) {
    std::ostringstream os;
    os << "<Context instance at " << std::hex << this->context_ << std::dec << ">";
    return os.str();
}

// Destructor
Context::~Context(void) {
    // free if the context is not a primary context and reference count goes to zero
    if (this->context_ != 0) {
        Context::m_.lock();
        if (--Context::shared_attributes_[this->context_].reference_count == 0) {
            Context::shared_attributes_.erase(this->context_);
            hipCtxDestroy(reinterpret_cast<hipCtx_t>(this->context_));
        }
        Context::m_.unlock();
    }
}

// --------------------------------------------------------------------------------------------------------------------
// PrimaryContext
// --------------------------------------------------------------------------------------------------------------------

}  // namespace merlin::cuda
