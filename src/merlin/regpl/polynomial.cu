// Copyright 2024 quocdang1998
#include "merlin/regpl/polynomial.hpp"

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// Polynomial
// ---------------------------------------------------------------------------------------------------------------------

// Copy data to a pre-allocated memory
void * regpl::Polynomial::copy_to_gpu(regpl::Polynomial * gpu_ptr, void * coeff_data_ptr,
                                      std::uintptr_t stream_ptr) const {
    // initialize buffer to store data of the copy before cloning it to GPU
    regpl::Polynomial copy_on_gpu;
    // shallow copy of coefficients, orders and term index
    double * coeff_ptr = reinterpret_cast<double *>(coeff_data_ptr);
    copy_on_gpu.coeff_.data() = coeff_ptr;
    copy_on_gpu.coeff_.size() = this->size();
    copy_on_gpu.order_ = this->order_;
    copy_on_gpu.ndim_ = this->ndim_;
    // copy data of each vector
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    ::hipMemcpyAsync(coeff_ptr, this->coeff_.data(), this->size() * sizeof(double), ::hipMemcpyHostToDevice, stream);
    // copy temporary object to GPU
    ::hipMemcpyAsync(gpu_ptr, &copy_on_gpu, sizeof(regpl::Polynomial), ::hipMemcpyHostToDevice, stream);
    // nullify pointer of temporary object to avoid de-allocate GPU pointer
    copy_on_gpu.coeff_.data() = nullptr;
    return reinterpret_cast<void *>(coeff_ptr + this->size());
}

// Copy data from GPU to CPU
void * regpl::Polynomial::copy_from_gpu(double * data_from_gpu, std::uintptr_t stream_ptr) noexcept {
    ::hipStream_t stream = reinterpret_cast<::hipStream_t>(stream_ptr);
    ::hipMemcpyAsync(this->coeff_.data(), data_from_gpu, this->size() * sizeof(double), ::hipMemcpyDeviceToHost,
                      stream);
    return reinterpret_cast<void *>(data_from_gpu + this->size());
}

}  // namespace merlin
