// Copyright quocdang1998
#include "merlin/env.hpp"

#include <hip/hip_runtime.h>  // ::hipCtxGetCurrent, ::hipGetDeviceCount, ::hipInit

#include "merlin/logger.hpp"

namespace merlin {

// Initialize CUDA context
void initialize_cuda_context(void) {
    // check for number of GPU
    ::hipError_t err_ = static_cast<::hipError_t>(::hipInit(0));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Initialize CUDA failed with error \"%s\"\n", ::hipGetErrorString(err_));
    }
    int num_gpu;
    err_ = static_cast<::hipError_t>(::hipGetDeviceCount(&num_gpu));
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get device failed with error \"%s\"\n", ::hipGetErrorString(err_));
    }
    // return empty context if no GPU was found
    if (num_gpu == 0) {
        WARNING("No GPU was found. Return empty CUDA context (GPU functions will have no effect).\n");
        return;
    }
    // try to get current context
    ::hipCtx_t current_ctx;
    err_ = static_cast<::hipError_t>(::hipCtxGetCurrent(&current_ctx));
    if ((err_ != 3) && (err_ != 0)) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    // initialized case (return success)
    if ((err_ == 0) && (current_ctx != nullptr)) {
        return;
    }
    // uninitialized or undefined
    std::printf("Initializing CUDA primary contexts.\n");
    for (int i_gpu = 0; i_gpu < num_gpu; i_gpu++) {
        err_ = static_cast<::hipError_t>(::hipDevicePrimaryCtxRetain(&current_ctx, i_gpu));
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "Retain primary context for GPU of ID %d failed with message \"%s\".\n",
                    i_gpu, ::hipGetErrorString(err_));
        }
        Environment::primary_contexts[i_gpu] = reinterpret_cast<std::uint64_t>(current_ctx);
    }
    // set back to default device
    ::hipCtxPushCurrent(reinterpret_cast<::hipCtx_t>(Environment::primary_contexts[Environment::default_gpu]));
}

// Destroy CUDA primary contexts
void destroy_cuda_context(void) {
    int num_gpu;
    ::hipGetDeviceCount(&num_gpu);
    // uninitialized or undefined
    for (int i_gpu = 0; i_gpu < num_gpu; i_gpu++) {
        ::hipDevicePrimaryCtxRelease(i_gpu);
        Environment::primary_contexts.erase(i_gpu);
    }
}

// Deallocate all pointers in deferred pointer array
void Environment::flush_cuda_deferred_deallocation(void) {
    Environment::mutex.lock();
    for (auto & [gpu, pointer] : Environment::deferred_gpu_pointer) {
        ::hipCtx_t gpu_context = reinterpret_cast<::hipCtx_t>(Environment::primary_contexts[gpu]);
        ::hipCtxPushCurrent(gpu_context);
        ::hipFree(pointer);
        ::hipCtxPopCurrent(&gpu_context);
    }
    Environment::deferred_gpu_pointer.clear();
    Environment::mutex.unlock();
}

}  // namespace merlin
