// Copyright quocdang1998
#include "merlin/env.hpp"

#include <hip/hip_runtime.h>  // ::hipCtxGetCurrent, ::hipGetDeviceCount, ::hipInit

#include "merlin/logger.hpp"    // FAILURE
#include "merlin/platform.hpp"  // __MERLIN_LINUX__, __MERLIN_WINDOWS__

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// CUDA environment
// ---------------------------------------------------------------------------------------------------------------------

// Initialize CUDA context
void initialize_cuda_context(void) {
    int num_gpu;
    ::hipError_t err_ = ::hipGetDeviceCount(&num_gpu);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get device failed with error \"%s\"\n", ::hipGetErrorString(err_));
    }
    // return empty context if no GPU was found
    if (num_gpu == 0) {
        WARNING("No GPU was found. Return empty CUDA context (GPU functions will have no effect).\n");
        return;
    }
    // try to get current context
    ::hipCtx_t current_ctx;
    err_ = static_cast<::hipError_t>(::hipCtxGetCurrent(&current_ctx));
    if ((err_ != 3) && (err_ != 0)) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    // initialized case (return success)
    if ((err_ == 0) && (current_ctx != nullptr)) {
        return;
    }
}

// Alarm for CUDA error
void alarm_cuda_error(void) {
    // check for any CUDA error
    ::hipError_t err_ = ::hipPeekAtLastError();
    if (err_ != 0) {
        WARNING("A CUDA error has occurred somewhere in the program with message \"%s\"", ::hipGetErrorString(err_));
    }
}

}  // namespace merlin
