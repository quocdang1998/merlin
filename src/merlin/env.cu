// Copyright quocdang1998
#include "merlin/env.hpp"

#include <hip/hip_runtime.h>  // ::hipCtxGetCurrent, ::hipGetDeviceCount, ::hipInit

#include "merlin/logger.hpp"  // FAILURE
#include "merlin/platform.hpp"  // __MERLIN_LINUX__, __MERLIN_WINDOWS__

namespace merlin {

// Initialize CUDA context
void initialize_cuda_context(void) {
    int num_gpu;
    ::hipError_t err_ = ::hipGetDeviceCount(&num_gpu);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get device failed with error \"%s\"\n", ::hipGetErrorString(err_));
    }
    // return empty context if no GPU was found
    if (num_gpu == 0) {
        WARNING("No GPU was found. Return empty CUDA context (GPU functions will have no effect).\n");
        return;
    }
    // try to get current context
    ::hipCtx_t current_ctx;
    err_ = static_cast<::hipError_t>(::hipCtxGetCurrent(&current_ctx));
    if ((err_ != 3) && (err_ != 0)) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    // initialized case (return success)
    if ((err_ == 0) && (current_ctx != nullptr)) {
        return;
    }
    // uninitialized or undefined
    for (int i_gpu = 0; i_gpu < num_gpu; i_gpu++) {
        err_ = ::hipSetDevice(i_gpu);
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "Initialize primary context for GPU of ID %d failed with message \"%s\".\n",
                    i_gpu, ::hipGetErrorString(err_));
        }
        ::hipCtxGetCurrent(&current_ctx);
        Environment::primary_contexts[i_gpu] = reinterpret_cast<std::uintptr_t>(current_ctx);
    }
    // set back to default device
    ::hipSetDevice(Environment::default_gpu);
}

// Deallocate all pointers in deferred pointer array
void Environment::flush_cuda_deferred_deallocation(void) {
    Environment::mutex.lock();
    for (auto & [gpu, pointer] : Environment::deferred_gpu_pointer) {
        ::hipCtx_t gpu_context = reinterpret_cast<::hipCtx_t>(Environment::primary_contexts[gpu]);
        ::hipCtxPushCurrent(gpu_context);
        ::hipFree(pointer);
        ::hipCtxPopCurrent(&gpu_context);
    }
    Environment::deferred_gpu_pointer.clear();
    Environment::mutex.unlock();
}

}  // namespace merlin
