// Copyright quocdang1998
#include "merlin/env.hpp"

#include <hip/hip_runtime.h>  // ::hipInit

#include "merlin/color.hpp"     // merlin::cout_terminal, merlin::cuprintf_terminal
#include "merlin/logger.hpp"    // merlin::Fatal, merlin::Warning

namespace merlin {

// ---------------------------------------------------------------------------------------------------------------------
// CUDA environment
// ---------------------------------------------------------------------------------------------------------------------

// Initialize CUDA context
void initialize_cuda_context(void) {
    // initialize context
    ::hipInit(0);
    // get number of CUDA capable GPUs
    int num_gpu;
    ::hipError_t err_ = ::hipGetDeviceCount(&num_gpu);
    if (err_ != 0) {
        Fatal<cuda_runtime_error>("Get device failed with error \"%s\"\n", ::hipGetErrorString(err_));
    }
    // return empty context if no GPU was found
    if (num_gpu == 0) {
        Warning("No GPU was found. Return empty CUDA context (GPU functions will have no effect).\n");
        return;
    }
}

// Alarm for CUDA error
void alarm_cuda_error(void) {
    // check for any CUDA error
    ::hipError_t err_ = ::hipPeekAtLastError();
    if (err_ != 0) {
        Warning("A CUDA error has occurred somewhere in the program with message \"%s\"", ::hipGetErrorString(err_));
    }
}

}  // namespace merlin
