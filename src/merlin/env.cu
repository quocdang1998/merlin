// Copyright quocdang1998
#include "merlin/env.hpp"

#include <hip/hip_runtime.h>  // ::hipCtxGetCurrent

#include "merlin/logger.hpp"

namespace merlin {

// Initialize CUDA context
void initialize_cuda_context(void) {
    // check for number of GPU
    int num_gpu;
    ::hipError_t err_ = ::hipGetDeviceCount(&num_gpu);
    if (err_ != 0) {
        FAILURE(cuda_runtime_error, "Get device failed with error \"%s\"\n", ::hipGetErrorString(err_));
    }
    // return empty context if no GPU was found
    if (num_gpu == 0) {
        WARNING("No GPU was found. Return empty CUDA context (GPU functions will have no effect).\n");
        return;
    }
    // try to get current context
    ::hipCtx_t current_ctx;
    err_ = static_cast<::hipError_t>(::hipCtxGetCurrent(&current_ctx));
    if ((err_ != 3) && (err_ != 0)) {
        FAILURE(cuda_runtime_error, "Get current context failed with message \"%s\".\n", ::hipGetErrorString(err_));
    }
    // initialized case (return success)
    if ((err_ == 0) && (current_ctx != nullptr)) {
        return;
    }
    // uninitialized or undefined
    for (int i_gpu = 0; i_gpu < num_gpu; i_gpu++) {
        err_ = ::hipSetDevice(i_gpu);
        if (err_ != 0) {
            FAILURE(cuda_runtime_error, "Set current GPU of ID %d failed with message \"%s\".\n",
                    i_gpu, ::hipGetErrorString(err_));
        }
        ::hipCtxGetCurrent(&current_ctx);
        Environment::primary_contexts[i_gpu] = reinterpret_cast<std::uint64_t>(current_ctx);
    }
    // set back to default device
    ::hipSetDevice(Environment::default_gpu);
}

}  // namespace merlin
